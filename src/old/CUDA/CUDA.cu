#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "TypeDef.h"
#include "WordsRange.h"
#include "CUDA.cuh"
#include "TargetPhraseDev.cuh"
#include "WordsRangeDev.cuh"
#include "WordsBitmapDev.cuh"
#include "Hypothesis.cuh"
#include "Stack.cuh"
#include "Stacks.cuh"
#include "Debug.cuh"
#include "TypeDef.cuh"

using namespace std;

template<typename T>
class Set
{
protected:
	size_t m_maxSize, m_size;
	T *m_array;
public:
	__device__ Set(size_t maxSize)
	{ 
	  m_maxSize = maxSize;
	  m_size = 0;
	  m_array = new T[maxSize];
	}

	__device__ ~Set()
	{
		delete m_array;
	}
	
	__device__ bool Add(const T &obj)
	{
		if (m_size >= m_maxSize) {
			return false;
		}
		
		m_array[m_size] = obj;
		++m_size;
		return true;
	}
};

///////////////////////////////////////////////////////////
__global__ void InitGPU()
{
	debugStrSize = 0;
	
	char *strSource = "Sample string";
	AppendStr(debugStr, debugStrSize, strSource);

	g_emptyTP = new TargetPhraseDev();
	g_emptyTP->SetEmpty();
}

void InitGPU(const std::vector<SCORE> &weights)
{	

	InitGPU<<<1, 1>>>();

	UINT32 numScores = weights.size();

 	CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(g_numScoresDev), &numScores, sizeof(UINT32), 0, hipMemcpyHostToDevice));

	SCORE *weightDevArr;
	CALL_CUDA(hipMalloc((SCORE**)&weightDevArr, sizeof(SCORE) * numScores));
 	CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(g_weightsDev), &weightDevArr, sizeof(SCORE*), 0, hipMemcpyHostToDevice));

	SCORE weightHost[numScores];
	std::copy(weights.begin(), weights.end(), weightHost);
 	CALL_CUDA(hipMemcpy(weightDevArr, &weightHost, sizeof(SCORE) * numScores, hipMemcpyHostToDevice));

 	DebugScores<<<1,1>>>();
}

void FinalizeGPU()
{
	cerr << "Shutting down" << endl;

	UINT32 debugStrSizeHost;
 	CALL_CUDA(hipMemcpyFromSymbol(&debugStrSizeHost, HIP_SYMBOL(debugStrSize), sizeof(UINT32), 0, hipMemcpyDeviceToHost));
	cerr << "debugStrSizeHost=" << debugStrSizeHost << endl;

	char debugStrHost[DEBUG_STR_MAX];
 	CALL_CUDA(hipMemcpyFromSymbol(&debugStrHost, HIP_SYMBOL(debugStr), debugStrSizeHost, 0, hipMemcpyDeviceToHost));
	cerr << "debugStrHost=" << debugStrHost << endl;

	UINT32 numScoresHost;
 	CALL_CUDA(hipMemcpyFromSymbol(&numScoresHost, HIP_SYMBOL(g_numScoresDev), sizeof(UINT32), 0, hipMemcpyDeviceToHost));
	cerr << "numScoresHost=" << numScoresHost << endl;

	SCORE tempHost;
 	CALL_CUDA(hipMemcpyFromSymbol(&tempHost, HIP_SYMBOL(temp), sizeof(SCORE), 0, hipMemcpyDeviceToHost));
	cerr << "tempHost=" << tempHost << endl;
	
}

void InitInputInfo(size_t inputSize, size_t numPaths)
{	
	g_inputInfoHost.inputSize = inputSize;
	g_inputInfoHost.numPaths = numPaths;

 	CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(g_inputInfoDev), &g_inputInfoHost, sizeof(g_inputInfoHost), 0, hipMemcpyHostToDevice));

 	g_pathInfoHost = (PathInfo*) malloc(sizeof(PathInfo) * numPaths);

	DebugInputInfo<<<1,1>>>();
}

void SetTargetPhrases(size_t pathInd, void *tpsMemHost, size_t memSize, const FastMoses::WordsRange &range)
{
	PathInfo &pathInfo = g_pathInfoHost[pathInd];
	pathInfo.range.startPos = range.startPos;
	pathInfo.range.endPos = range.endPos;
	
	void *tpsMemDev = NULL;
	if (tpsMemHost) {
	
		UINT32 *tpsMemHostUINT32 = (UINT32*) tpsMemHost;
		UINT32 numTPS = tpsMemHostUINT32[0];
		pathInfo.numTPS = numTPS;
		
		// target phrases to device		
		CALL_CUDA(hipMalloc((void**)&tpsMemDev, memSize));	
	  	CALL_CUDA(hipMemcpy(tpsMemDev, tpsMemHost, memSize, hipMemcpyHostToDevice));
	}
	else {
		pathInfo.numTPS = 0;
	}
	
	pathInfo.tpsMem = tpsMemDev;	
}

__global__ void CreateTPSDev()
{
	// initialise tps
	UINT32 numPaths = g_inputInfoDev.numPaths;
	char str[111];
	itoa(numPaths, str, 10);

	DebugStr(" numPaths=");
	DebugStr(str);
	DebugStr("\n");

	for (size_t pathInd = 0; pathInd < numPaths; ++pathInd) {
  		PathInfo &path = g_pathInfoDev[pathInd];
  		UINT32 numTPS = path.numTPS;
  		void *tpsMem = path.tpsMem;
		DebugCreateTPSDev(pathInd, numTPS, tpsMem);

		size_t tempMem = (size_t) tpsMem;
		tempMem += sizeof(UINT32);
		tpsMem = (void*) tempMem;

		//numTPS = 1;
  		path.tps = (TargetPhraseDev*) malloc(sizeof(TargetPhraseDev) * numTPS);
		for (size_t tpInd = 0; tpInd < numTPS; ++tpInd) {
			TargetPhraseDev &tp = path.tps[tpInd];

			/*
			AppendStr(debugStr, debugStrSize, " tpsMem=");
			itoa((int)tpsMem, str, 10);
			AppendStr(debugStr, debugStrSize, str);
			AppendStr(debugStr, debugStrSize, " ");
			*/
			tpsMem = tp.Initialize(tpsMem);
			/*
			itoa((int)tpsMem, str, 10);
			AppendStr(debugStr, debugStrSize, str);
			AppendStr(debugStr, debugStrSize, " ");

			DebugCreateTPSDev2(tpInd, (size_t) tpsMem, tp);
			*/
		}

  	}
}

void CompleteInputInfo()
{
	UINT32 numPaths = g_inputInfoHost.numPaths;
	size_t memSize = sizeof(PathInfo) * numPaths;
	 
	PathInfo *pathInfoDevArr;
	CALL_CUDA(hipMalloc((PathInfo**)&pathInfoDevArr, memSize));
 	CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(g_pathInfoDev), &pathInfoDevArr, sizeof(PathInfo*), 0, hipMemcpyHostToDevice));

 	CALL_CUDA(hipMemcpy(pathInfoDevArr, g_pathInfoHost, memSize, hipMemcpyHostToDevice));

 	DebugTPS<<<1,1>>>();

 	CreateTPSDev<<<1,1>>>();

}

__global__ void InitStacks(size_t inputSize)
{
	g_stacks = new Stacks(inputSize);
	
	Stack &firstStack = g_stacks->GetStack(0);
	
	Hypothesis emptyHypo(inputSize, *g_emptyTP);
	firstStack.AddHypo(emptyHypo);

  	// init todo hypos 
	g_hypos = new Hypothesis[NUM_NEW_HYPOS];

	DebugStr("NUM_NEW_HYPOS=");
	DebugStr(NUM_NEW_HYPOS);
	DebugStr(" ");
	DebugStr(inputSize);
	DebugStr(" ");
	DebugStr((int) g_hypos);
	DebugStr("\n");
}

void InitStacksHost(size_t inputSize)
{
	InitStacks<<<1, 1>>>(inputSize);
}

__global__ void InitStack(size_t stackInd)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row > 0) {
  	return;
  }
  
  DebugStr("g_hypos=");
  DebugStr((int) g_hypos);
  DebugStr("\n");

  // create all potential new hypos in hypos array
  UINT32 numPaths = g_inputInfoDev.numPaths;

  Stack &stack = g_stacks->GetStack(stackInd);
  
  g_numHypos = 0;
  for (size_t prevHypoInd = 0; prevHypoInd < stack.GetNumHypos(); ++prevHypoInd) {
  	const Hypothesis &prevHypo = stack.GetHypo(prevHypoInd);
    	
  	for (size_t pathInd = 0; pathInd < numPaths; ++pathInd) {
  		PathInfo &path = g_pathInfoDev[pathInd];
  		const WordsRangeDev &range = path.range;
  		void *tpsMem = path.tpsMem;
  		
  		if (tpsMem) {
			UINT32 *tpsUINT32 = (UINT32*) tpsMem;
			UINT32 numTPS = tpsUINT32[0];
			
	  		for (size_t tpInd = 0; tpInd < numTPS; ++tpInd) {
	  			const TargetPhraseDev &tp = path.tps[tpInd];
	  			
		  		Hypothesis &newHypo = g_hypos[g_numHypos];
		  		newHypo.SetHypo(prevHypo, tp, range);


		  		DebugStr((int) &newHypo);
		  		DebugStr("=");
		  		DebugBoolToStr(newHypo.IsValid());
		  		DebugStr("\n");

		  	  	++g_numHypos;
		  	}
	  	}
  	}
  }

	char str[111];

	DebugStr(" g_numHypos=");
	itoa(g_numHypos, str, 10);
	AppendStr(debugStr, debugStrSize, str);
}

void InitStackHost(size_t stackInd)
{
	InitStack<<<4, 1>>>(stackInd);
}

__device__ void ProcessHypo(size_t hypoInd)
{
	Hypothesis &hypo = g_hypos[hypoInd];
	hypo.Process(g_numScoresDev, g_weightsDev);

	//Set<int> s(3);
	//s.Add(4);
	//S s(3);
	
}

__global__ void ProcessStackParallel(size_t stackInd)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= g_numHypos) {
		return;
	}
	
	ProcessHypo(row);
}

__global__ void ProcessStackSerial(size_t stackInd)
{
	for (size_t i = 0; i < g_numHypos; ++i) {
		ProcessHypo(i);
	}
}

void ProcessStackHost(size_t stackInd)
{
	//ProcessStackParallel<<<4, 999>>>(stackInd);
	ProcessStackSerial<<<1, 999>>>(stackInd);
}

__global__ void FinalizeStack(size_t stackInd)
{
	size_t numAdded = 0;
	for (size_t i = 0; i < g_numHypos; ++i) {
		Hypothesis &hypo = g_hypos[i];

		DebugStr("hello");
		DebugStr((int) &hypo);
		DebugBoolToStr(hypo.IsValid());
		DebugStr(hypo.GetBitmap().GetNumWordsCovered());

		if (hypo.IsValid()) {
			DebugStr("goodbye");
			const WordsBitmapDev &bm = hypo.GetBitmap();
			int numWordsCovered = bm.GetNumWordsCovered();
			Stack &destStack = g_stacks->GetStack(numWordsCovered);
			destStack.AddHypo(hypo);

			++numAdded;
		}
	}

	char str[111];

	DebugStr(" numAdded=");
	itoa(numAdded, str, 10);
	AppendStr(debugStr, debugStrSize, str);

}

void FinalizeStackHost(size_t stackInd)
{
	FinalizeStack<<<1,1>>>(stackInd);
}

__global__ void DebugStacksDev(size_t inputSize)
{
	char str[111];

	DebugStr("Stacks=");

	for (size_t i = 0; i <= inputSize; ++i) {
		Stack &stack = g_stacks->GetStack(i);
		size_t numHypos = stack.GetNumHypos();

		itoa(numHypos, str, 10);
		AppendStr(debugStr, debugStrSize, str);
		DebugStr(" ");
	}

	DebugStr("\n");

}


void DebugStacks(size_t inputSize)
{
	DebugStacksDev<<<1,1>>>(inputSize);
	
}


