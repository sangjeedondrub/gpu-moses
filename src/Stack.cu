#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"
#include "CUDA/Util.h"

using namespace std;


Stack::Stack(const Manager &mgr)
:m_coll()
{
  //mgr.system.params.
  m_coll.GetVec().Reserve(5000);

  hipDeviceSynchronize();
  //cerr << "m_arr=" << m_arr << endl;

  //debugStr[0] = 0x0;
}

__host__
Stack::~Stack()
{
  for (size_t i = 0; i < m_coll.size(); ++i) {
    Hypothesis *hypo = m_coll.GetVec()[i];
    hipFree(hypo);
  }
}

__device__
void Stack::add(Hypothesis *hypo)
{
  /*
  StrCat(debugStr, "hypo=");
  StrCat(debugStr, itoaDevice((size_t) hypo));
  */
  thrust::pair<bool, size_t> upper = m_coll.upperBound(hypo);
  if (upper.first) {
    // same hypo exist
    const Hypothesis *otherHypo = m_coll.GetVec()[upper.second];

    SCORE newScore = hypo->getFutureScore();
    SCORE otherScore = otherHypo->getFutureScore();

    if (newScore > otherScore) {
      // new hypo is better
      /*
      StrCat(debugStr, " ADDED winner=");

      char str[500];

      str[0] = 0x0;
      hypo->Debug(str);
      StrCat(debugStr,  str);

      StrCat(debugStr,  "\nloser=");
      str[0] = 0x0;
      otherHypo->Debug(str);
      StrCat(debugStr,  str);

      //StrCat(debugStr,  itoaDevice((size_t) otherHypo));
      StrCat(debugStr, "\n");
      */
      delete otherHypo;
      m_coll.GetVec()[upper.second] = hypo;
    }
    else {
      // existing hypo is better
      //StrCat(debugStr, " not ADDED\n");
      delete hypo;
    }
  }
  else {
    //StrCat(debugStr, " ADDED\n");
    m_coll.insert(hypo);
    //(*m_arr)[m_size] = hypo;
  }
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "stack size=" << size << endl;
  for (size_t i = 0; i < size; ++i) {
    //cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = m_coll.GetVec()[i];
    //cerr << "HH2:" << hypo << endl;

    //cerr << "HH3:" << h_s << endl;
    cerr << "hypo= " << hypo->Debug() << endl;
  }

  return strm.str();
}
