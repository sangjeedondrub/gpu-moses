#include "hip/hip_runtime.h"
#include <sstream>
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "TargetPhrase.h"
#include "Manager.h"
#include "System.h"
#include "CUDA/Util.h"

using namespace std;

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:mgr(&mgr)
,bitmap(mgr.GetInput().size())
,scores(mgr.system.featureFunctions.totalNumScores)
,stateData(mgr.system.featureFunctions.totalStateSize)
{
  sss = 453.54;
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  this->mgr = &mgr;
  prevHypo = NULL;

  path = &mgr.initPath;
  targetPhrase = &mgr.initPhrase;

  bitmap.Init();

  currTargetWordsRange.startPos = -1;
  currTargetWordsRange.endPos = -1;
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const InputPath &path)
{
  this->mgr = &mgr;
  this->prevHypo = &prevHypo;

  this->path = &path;
  targetPhrase = &tp;

  const Bitmap &prevBM = prevHypo.bitmap;
  bitmap.Init(prevBM, path.range);

  currTargetWordsRange.startPos = prevHypo.currTargetWordsRange.endPos + 1;
  currTargetWordsRange.endPos = prevHypo.currTargetWordsRange.endPos + tp.size();

  scores.PlusEqual(mgr.system, tp.GetScores());
  scores.PlusEqual(mgr.system, prevHypo.scores);

  mgr.system.featureFunctions.EvaluateWhenApplied(mgr, *this);
}

__device__
SCORE Hypothesis::getFutureScore() const
{
  return scores.GetTotal();
}

__device__
int Hypothesis::RecombineCompare(const Hypothesis &other) const
{
  // -1 = this < compare
  // +1 = this > compare
  // 0  = this ==compare
  int comp = bitmap.Compare(other.bitmap);
  if (comp != 0)
    return comp;

  return stateData.Compare(other.stateData);
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  output = hypo.getFutureScore();
  //output = hypo.sss; //score;
  //output = 456.789;
}

__host__
SCORE Hypothesis::GetFutureScore() const
{
    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*this, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    hipFree(d_s);

    return h_s;
}

__device__
VOCABID Hypothesis::GetCurrWord(size_t pos) const
{
  return (*targetPhrase)[pos];
}

/** recursive - pos is relative from start of sentence */
__device__
VOCABID Hypothesis::GetWord(size_t pos) const
{
  const Hypothesis *hypo = this;
  while (pos < hypo->currTargetWordsRange.startPos) {
    hypo = hypo->prevHypo;

    if (hypo == NULL) {
      __threadfence();         // ensure store issued before trap
      asm("trap;");
    }
  }
  return hypo->GetCurrWord(pos - hypo->currTargetWordsRange.startPos);
}


///////////////////////////////////////////////////////////////
__host__
std::string Hypothesis::Debug() const
{
  stringstream strm;
  strm << (int) (size_t) this << " ";

  char *d_str;
  hipMallocManaged(&d_str, 1000);
  hipDeviceSynchronize();
  hipMemset(d_str, 0, 1000);

  debugObj<<<1,1>>>(*this, d_str);
  hipDeviceSynchronize();

  strm << d_str;

  hipFree(d_str);

  return strm.str();;
}

__device__
void Hypothesis::Debug(char *out, size_t indent) const
{
  for (size_t i = 0; i < indent; ++i) {
    StrCat(out, "  ");
  }

  path->range.Debug(out);
  StrCat(out, "->");
  currTargetWordsRange.Debug(out);
  StrCat(out, " ");

  bitmap.Debug(out);
  StrCat(out, " ");

  scores.Debug(out);

  StrCat(out, "state=");
  for (size_t i = 0; i < stateData.size(); ++i) {
    unsigned int c = stateData[i];
    char *str = itoaDevice(c);
    StrCat(out, str);
    StrCat(out, " ");
  }

  if (prevHypo) {
     StrCat(out, "\n");
    prevHypo->Debug(out, indent + 1);
  }
}

