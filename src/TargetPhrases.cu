#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrases.h"
#include "CUDA/Util.h"

using namespace std;

TargetPhrases::TargetPhrases()
:m_vec(0)
{
}

TargetPhrases::~TargetPhrases()
{
  for (size_t i = 0; i < m_vec.size(); ++i) {
    const TargetPhrase *tp = m_vec[i];
    delete tp;
  }
}

void TargetPhrases::Add(const TargetPhrase *tp)
{
	m_vec.PushBack(tp);
}

__host__ std::string TargetPhrases::Debug() const
{
  stringstream strm;

  for (size_t i = 0; i < m_vec.size(); ++i) {
    const TargetPhrase *tp = m_vec[i];
    strm << tp->Debug() << endl;
  }
  return strm.str();
}

__global__ void checkTargetPhrases(char *str, const TargetPhrases &tps)
{
  size_t size = tps.size();
  char *tmp = itoaDevice(size);

  StrCpy(str, tmp);
}

