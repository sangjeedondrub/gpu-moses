/* Copyright (C) 1989, 1990, 1991, 1992 Free Software Foundation, Inc.
     Written by James Clark (jjc@jclark.com)

This file is part of groff.

groff is free software; you can redistribute it and/or modify it under
the terms of the GNU General Public License as published by the Free
Software Foundation; either version 2, or (at your option) any later
version.

groff is distributed in the hope that it will be useful, but WITHOUT ANY
WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
for more details.

You should have received a copy of the GNU General Public License along
with groff; see the file COPYING.  If not, write to the Free Software
Foundation, 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA. */
#include <string.h>
#include <hip/hip_runtime.h>

#define INT_DIGITS 19		/* enough for 64 bit integer */

__device__
char *itoaDevice(int i)
{
  /* Room for INT_DIGITS digits, - and '\0' */
  static char buf[INT_DIGITS + 2];
  char *p = buf + INT_DIGITS + 1;	/* points to terminating '\0' */
  if (i >= 0) {
    do {
      *--p = '0' + (i % 10);
      i /= 10;
    } while (i != 0);
    return p;
  }
  else {			/* i < 0 */
    do {
      *--p = '0' - (i % 10);
      i /= 10;
    } while (i != 0);
    *--p = '-';
  }
  return p;
}

__device__
size_t strlenDevice(const char *str)
{
  register const char *s;

  for (s = str; *s; ++s);
  return(s - str);
}


__device__ void MemCpy(char *dest, const char *src, size_t count)
{
  for (size_t i = 0; i < count; ++i) {
    dest[i] = src[i];
  }
}

__device__ void StrCpy(char *dest, const char *src)
{
  size_t len = strlenDevice(src);
  MemCpy(dest, src, len + 1);
}

__device__ char *StrCat(char *dest, const char *src)
{
  StrCpy(&dest[strlenDevice(dest)], src);
  return dest;
}

__device__
static const double powers_of_10[] = { 1, 10, 100, 1000, 10000, 100000, 1000000,
    10000000, 100000000, 1000000000 };

__device__
static void strreverse(char* begin, char* end)
{
    char aux;
    while (end > begin)
        aux = *end, *end-- = *begin, *begin++ = aux;
}

__device__
size_t modp_dtoa(double value, char* str, int prec)
{
    /* Hacky test for NaN
   * under -fast-math this won't work, but then you also won't
   * have correct nan values anyways.  The alternative is
   * to link with libmath (bad) or hack IEEE double bits (bad)
   */
    if (!(value == value)) {
        str[0] = 'n';
        str[1] = 'a';
        str[2] = 'n';
        str[3] = '\0';
        return (size_t)3;
    }
    /* if input is larger than thres_max, revert to exponential */
    const double thres_max = (double)(0x7FFFFFFF);

    double diff = 0.0;
    char* wstr = str;

    if (prec < 0) {
        prec = 0;
    } else if (prec > 9) {
        /* precision of >= 10 can lead to overflow errors */
        prec = 9;
    }

    /* we'll work in positive values and deal with the
     negative sign issue later */
    int neg = 0;
    if (value < 0) {
        neg = 1;
        value = -value;
    }

    // given 0.05, prec=1
    // whole = 0
    // tmp = (0.05)* 10 = 0.5
    // frac = 0
    // diff = tmp -frac == 0.5 - 0.0 = 0.5
    //
    int whole = (int)value;
    double tmp = (value - whole) * powers_of_10[prec];
    uint32_t frac = (uint32_t)(tmp);
    diff = tmp - frac;

    if (diff > 0.5) {
        ++frac;
        /* handle rollover, e.g.  case 0.99 with prec 1 is 1.0  */
        if (frac >= powers_of_10[prec]) {
            frac = 0;
            ++whole;
        }
    } else if (diff == 0.5 && prec > 0 && (frac & 1)) {
        /* if halfway, round up if odd, OR
       if last digit is 0.  That last part is strange */
        ++frac;
        if (frac >= powers_of_10[prec]) {
            frac = 0;
            ++whole;
        }
    } else if (diff == 0.5 && prec == 0 && (whole & 1)) {
        ++frac;
        if (frac >= powers_of_10[prec]) {
            frac = 0;
            ++whole;
        }
    }

    /* for very large numbers switch back to native sprintf for exponentials.
     anyone want to write code to replace this? */
    /*
     normal printf behavior is to print EVERY whole number digit
     which can be 100s of characters overflowing your buffers == bad
     */
    if (value > thres_max) {
        //sprintf(str, "%e", neg ? -value : value);
        //return strlen(str);
    }

    int count = prec;
    while (count > 0) {
        --count;
        *wstr++ = (char)(48 + (frac % 10));
        frac /= 10;
    }
    if (frac > 0) {
        ++whole;
    }

    /* add decimal */
    if (prec > 0) {
        *wstr++ = '.';
    }

    /* do whole part
   * Take care of sign conversion
   * Number is reversed.
   */
    do
        *wstr++ = (char)(48 + (whole % 10));
    while (whole /= 10);
    if (neg) {
        *wstr++ = '-';
    }
    *wstr = '\0';
    strreverse(str, wstr - 1);
    return (size_t)(wstr - str);
}

__device__ char *ftoaDevice(double value)
{
  static char buf[100];
  modp_dtoa(value, buf, 4);
  return buf;
}




