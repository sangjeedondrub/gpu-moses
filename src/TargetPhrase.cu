#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrase.h"
#include "MyVocab.h"
#include "System.h"
#include "CUDA/Util.h"

#include "TypeDef.h"

using namespace std;

TargetPhrase *TargetPhrase::CreateFromString(const System &sys, const std::string &str)
{
	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();
	vector<VOCABID> ids = vocab.GetOrCreateIds(str);
  //cerr << "ids=" << ids.size() << endl;

	TargetPhrase *tp = new TargetPhrase(sys, ids);
	return tp;
}

__host__
TargetPhrase::TargetPhrase(const System &sys, size_t size)
:Phrase(size)
,m_scores(sys.featureFunctions.totalNumScores)
{

}

TargetPhrase::TargetPhrase(const System &sys, const std::vector<VOCABID> &ids)
:Phrase(ids)
,m_scores(sys.featureFunctions.totalNumScores)
{
}

__host__ std::string TargetPhrase::Debug() const
{
  stringstream strm;
  strm << Phrase::Debug() << " Scores:" << m_scores.Debug();

  return strm.str();
}

__global__ void checkTargetPhrase(char *str, const TargetPhrase &phrase)
{
  VOCABID totVocabId;
  SCORE totScore;

  size_t size = phrase.size();
  totVocabId = size;
  for (size_t i = 0; i < size; ++i) {
    VOCABID id = phrase[i];
    totVocabId += id;
  }

  size = phrase.GetScores().size();
  totScore = size;
  for (size_t i = 0; i < size; ++i) {
      SCORE score = phrase.GetScores()[i];
      totScore += score;
  }

  char *tmp = itoaDevice(totVocabId + totScore);
  StrCpy(str, tmp);

}

