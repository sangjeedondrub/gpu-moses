#include "hip/hip_runtime.h"
#include <iostream>
#include "Manager.h"
#include "Phrase.h"
#include "Hypothesis.h"
#include "Stack.h"
#include "Range.h"
#include "System.h"
#include "MyVocab.h"
#include "FF/FeatureFunctions.h"
#include "TranslationModel/PhraseTableMemory.h"

using namespace std;

Manager::Manager(const System &sys, const std::string &inputStr)
:system(sys)
,m_tpsVec(0)
,initPhrase(sys, 0)
{
  m_input = Phrase::CreateFromString(inputStr);
  //cerr << "m_input=" << m_input->Debug() << endl;

}

Manager::~Manager()
{
  delete m_input;
}

__global__ void checkManager(char *str, const Manager &mgr)
{
  const Phrase &input = mgr.GetInput();
  checkPhrase<<<1,1>>>(str, input);

}


///////////////////////////////////////
__global__
void Lookup(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  const Phrase &input = mgr.GetInput();
  size_t inputSize = input.size();

  const PhraseTableMemory &pt = *mgr.system.featureFunctions.pt;
  const TargetPhrases *tps = pt.Lookup(input, start, end);

  mgr.GetInputPath(start, end).targetPhrases =  tps;
}

__global__
void Process1stStack(const Manager &mgr, Stacks &stacks)
{
  Hypothesis *hypo = new Hypothesis(mgr);
  hypo->Init(mgr);
  Stack &stack = stacks[0];
  stack.add(hypo);
}

__global__
void ProcessStack(size_t stackInd, const Manager &mgr, Stacks &stacks)
{
  int hypoInd = blockIdx.x;
  int start = blockIdx.y;
  int end = blockIdx.z;

  if (start > end) {
    return;
  }

  const InputPath &path = mgr.GetInputPath(start, end);
  const TargetPhrases *tps = path.targetPhrases;
  if (tps == NULL || tps->size() == 0) {
    return;
  }

  const Stack &stack = stacks[stackInd];
  const Vector<Hypothesis*> &vec = stack.getArr();
  const Hypothesis &prevHypo = *vec[hypoInd];
  const Bitmap &prevBM = prevHypo.bitmap;

  if (prevBM.Overlap(path.range)) {
    return;
  }

  for (size_t i = 0; i < tps->size(); ++i) {
    const TargetPhrase *tp = (*tps)[i];
    assert(tp);

    Hypothesis *hypo = new Hypothesis(mgr);
    hypo->Init(mgr, prevHypo, *tp, path);
    const Bitmap &newBM = hypo->bitmap;
    size_t wordsCovered = newBM.GetNumWordsCovered();

    Stack &destStack = stacks[wordsCovered];

    Lock &lock = destStack.getLock();
    lock.lock();

    destStack.add(hypo);

    lock.unlock();
  }
}

__global__
void GetBestHypo(const Manager &mgr, const Stack &lastStack, VOCABID *vocabIds)
{
  const Hypothesis *bestHypo = NULL;
  SCORE bestScore = -999999;

  const Vector<Hypothesis*> &hypos = lastStack.getArr();
  for (size_t i = 0; i < hypos.size(); ++i) {
    const Hypothesis *hypo = hypos[i];
    if (hypo->getFutureScore() > bestScore) {
      bestScore = hypo->getFutureScore();
      bestHypo = hypo;
    }
  }

  assert(bestHypo);
  size_t pos = 0;
  while (bestHypo) {
    const TargetPhrase &tp = *bestHypo->targetPhrase;

    for (size_t i = tp.size(); i > 0; --i) {
      VOCABID id = tp[i - 1];
      vocabIds[pos] = id;

      ++pos;
    }

    bestHypo = bestHypo->prevHypo;
  }
}
///////////////////////////////////////
__host__
void Manager::Process()
{
  cerr << endl;
  /*
  char *str;
  hipHostMalloc(&str, 10000);

  checkManager<<<1,1>>>(str, *this);
  hipDeviceSynchronize();
  cerr << "mgr=" << str << endl;
  */

  size_t inputSize = m_input->GetSize();
  cerr << "inputSize=" << inputSize << endl;
  InitInputPaths();

  Lookup<<<inputSize, inputSize>>>(*this);
  hipDeviceSynchronize();
  cerr << "tps=" << DebugTPSArr() << endl;

  m_stacks.Init(*this, m_input->GetSize() + 1);

  Process1stStack<<<1,1>>>(*this, m_stacks);
  hipDeviceSynchronize();

  Stack &stack = m_stacks.Get(0);
  cerr << "1st stack=" << stack.GetSize() << endl;

  for (size_t stackInd = 0; stackInd < inputSize; ++stackInd) {
    const Stack &stack = m_stacks.Get(stackInd);
    //cerr << "HH1:" << stack.debugStr << endl;

    size_t stackSize = stack.GetSize();
    //cerr << "HH2" << endl;

    //ProcessStack<<<1,1>>>(stackInd, *this, m_stacks);
    //ProcessStack<<<stackSize, 1>>>(stackInd, *this, m_stacks);
    //ProcessStack<<<1, inputSize>>>(stackInd, *this, m_stacks); // deadlock with lock
    //ProcessStack<<<stackSize, inputSize>>>(stackInd, *this, m_stacks);

    dim3 blocks(stackSize, inputSize, inputSize);
    ProcessStack<<<blocks, 1>>>(stackInd, *this, m_stacks);
    //cerr << "HH3" << endl;

    hipDeviceSynchronize();
    //cerr << "HH4" << endl;
    m_stacks.PrintStacks();
    //cerr << "stack=" << stack.Debug() << endl;
    //cerr << "HH6" << endl;
 }

  cerr << m_stacks.Back().Debug() << endl;

  Vector<VOCABID> bestHypo(100, NOT_FOUND_DEVICE);
  cerr << "before=" << bestHypo.Debug() << endl;
  hipDeviceSynchronize();

  GetBestHypo<<<1,1>>>(*this, m_stacks.Back(), bestHypo.data());
  hipDeviceSynchronize();

  cerr << "after=" << bestHypo.Debug() << endl;

  cerr << "Best Translation: ";
  for (size_t i = 0; i < bestHypo.size(); ++i) {
    VOCABID id = bestHypo[i];
    if (id == NOT_FOUND_DEVICE) {
      break;
    }
    //cerr << "id=" << id << " ";
    cerr << FastMoses::MyVocab::Instance().GetString(id) << " ";
  }
  cerr << endl;
}

std::string Manager::DebugTPSArr() const
{
  std::stringstream strm;
  for (size_t i = 0; i < m_tpsVec.size(); ++i) {
    const InputPath &path = m_tpsVec[i];
    const TargetPhrases *tps = path.targetPhrases;
    strm << path.range.Debug() << " ";
    strm << tps;
    if (tps) {
      strm << "(" << tps->GetSize() << ")";
    }
    strm << " " << endl;
  }
  return strm.str();
}

__global__
void InitPathRange(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  InputPath &path = mgr.GetInputPath(start, end);
  path.range.startPos = start;
  path.range.endPos = end;
}

__host__
void Manager::InitInputPaths()
{
  size_t inputSize = m_input->GetSize();
  m_tpsVec.Resize(inputSize * inputSize);

  InitPathRange<<<inputSize, inputSize>>>(*this);
  hipDeviceSynchronize();
}

__device__
size_t Manager::RangeToInd(int start, int end) const
{
  const Phrase &input = GetInput();
  size_t inputSize = input.size();

  size_t ret = start * inputSize + end;
  return ret;
}

__device__
InputPath &Manager::GetInputPath(int start, int end)
{
  InputPath &path = m_tpsVec[RangeToInd(start, end)];
  return path;
}

__device__
const InputPath &Manager::GetInputPath(int start, int end) const
{
  const InputPath &path = m_tpsVec[RangeToInd(start, end)];
  return path;
}
