#include "hip/hip_runtime.h"
#include "Distortion.h"
#include "../Hypothesis.h"
#include "../Range.h"
#include "../InputPath.h"
#include "../Manager.h"

Distortion::Distortion(size_t startInd, const std::string &line)
:StatefulFeatureFunction(startInd, line)
{
  classId = FeatureFunction::ClassId::Distortion;
	stateSize = sizeof(size_t);

	ReadParameters();
}

__device__
void Distortion::EvaluateWhenApplied(const Manager &mgr, Hypothesis &hypo) const
{

  if (hypo.prevHypo == NULL) {
    return;
  }

  // score
  const Range &prevRange = hypo.prevHypo->path->range;
  const Range &currRange = hypo.path->range;

  int dist = - ComputeDistortionDistance(prevRange, currRange);

  ScoresUnmanaged &scores = hypo.scores;

  //scores.PlusEqual(*this, currRange.startPos + currRange.endPos);
  scores.PlusEqual(mgr.system, *this, dist);

  const size_t &endPos = currRange.endPos;
  SetState(hypo, (const char*) &endPos);
}

__device__
int Distortion::ComputeDistortionDistance(const Range& prev,
    const Range& current) const
{
  int dist = 0;
  if (prev.GetNumWordsCovered() == 0) {
    dist = current.startPos;
  }
  else {
    dist = (int) prev.endPos - (int) current.startPos + 1;
  }
  return abs(dist);
}
