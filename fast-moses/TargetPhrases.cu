#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrases.h"
#include "CUDA/Util.h"

using namespace std;

TargetPhrases::TargetPhrases()
:m_vec(true, 0)
{
}

TargetPhrases::~TargetPhrases()
{
  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    const TargetPhrase *tp = m_vec.Get(i);
    delete tp;
  }
}

void TargetPhrases::Add(const TargetPhrase *tp)
{
	m_vec.PushBack(tp);
}

__host__ std::string TargetPhrases::Debug() const
{
  stringstream strm;

  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    const TargetPhrase *tp = m_vec.Get(i);
    strm << tp->Debug() << endl;
  }
  return strm.str();
}

__global__ void checkTargetPhrases(char *str, const TargetPhrases &tps)
{
  size_t size = tps.size();
  char *tmp = itoaDevice(size);

  StrCpy(str, tmp);
}

