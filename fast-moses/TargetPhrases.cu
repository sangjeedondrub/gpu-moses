#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrases.h"

using namespace std;

TargetPhrases::~TargetPhrases()
{
  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    const TargetPhrase *tp = m_vec.Get(i);
    delete tp;
  }
}

void TargetPhrases::Add(const TargetPhrase *tp)
{
	m_vec.push_back(tp);
}

__host__ std::string TargetPhrases::Debug() const
{
  stringstream strm;

  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    const TargetPhrase *tp = m_vec.Get(i);
    strm << tp->Debug() << endl;
  }
  return strm.str();
}

__global__ void checkTargetPhrases(size_t &tot, const TargetPhrases &tps)
{
  tot = tps.size();
}

