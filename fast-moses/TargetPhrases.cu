#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrases.h"
#include "itoa.h"

using namespace std;

TargetPhrases::~TargetPhrases()
{
  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    const TargetPhrase *tp = m_vec.Get(i);
    delete tp;
  }
}

void TargetPhrases::Add(const TargetPhrase *tp)
{
	m_vec.push_back(tp);
}

__host__ std::string TargetPhrases::Debug() const
{
  stringstream strm;

  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    const TargetPhrase *tp = m_vec.Get(i);
    strm << tp->Debug() << endl;
  }
  return strm.str();
}

__device__ void MemCpy(char *dest, const char *src, size_t count)
{
  for (size_t i = 0; i < count; ++i) {
    dest[i] = src[i];
  }
}

__device__ void StrCpy(char *dest, const char *src)
{
  size_t len = strlenDevice(src);
  MemCpy(dest, src, len + 1);
}
__global__ void checkTargetPhrases(char *str, const TargetPhrases &tps)
{
  size_t size = tps.size();
  char *tmp = itoaDevice(size);

  StrCpy(str, tmp);
}

