#include "hip/hip_runtime.h"
#include <array>
#include <iterator>
#include <iostream>
#include <thrust/binary_search.h>
#include "Test.h"
#include "Phrase.h"
#include "CUDA/Set.h"
#include "CUDA/Map.h"
#include "CUDA/Managed.h"
#include "CUDA/Vector.h"

using namespace std;

#define N 10
#define N_SOUGHT 2

//////////////////////////////////////////////////////

class C2 : public Managed
{
public:
	int i;
	int arr[N] = {2,4,6,8,10,12,14,16,18,20};
	int out[3];

	C2(int v)
	{
		i = v;
	}

	__device__ void Add(int v)
	{
		i += v;
	}

	void Search()
	{
		int sought[3] = {4, 5, 6};

	    thrust::binary_search(std::begin(arr), std::end(arr),
	    		std::begin(sought), std::end(sought),
	    		std::begin(out));

	}
};

__global__ void Temp(C2 &o)
{
  o.Add(2);
  //o.Search();
}

__global__ void KernelC2(C2 &o)
{
  o.Add(3);
  //Temp<<<2,1>>>(o);
}

void Test2()
{
  //KernelTCLass<<<1,1>>>();
  //KernelTAdd<<<1,1>>>(3);
  C2 *oHeap = new C2(6);
  hipDeviceSynchronize();
  cerr << "oHeap=" << oHeap->i << endl;

  KernelC2<<<1,1>>>(*oHeap);
  hipDeviceSynchronize();
  cerr << "oHeap=" << oHeap->i << endl;

  Temp<<<2,1>>>(*oHeap);
  //oHeap->Add<<<1,1>>>(4);

  oHeap->Search();
  for (size_t i = 0; i < 3; ++i) {
	  cerr << oHeap->out[i] << " ";
  }
  cerr << endl;

  delete oHeap;

  C2 oStack(7);
  cerr << "oStack=" << oStack.i << endl;
  //oStack.Add(4);
  //cerr << "oStack=" << oStack.i << endl;

}

//////////////////////////////////////////////////////

void Test3()
{
  Phrase *input = Phrase::CreateFromString("dast ist eine kleines haus");
  cerr << "input=" << input->Debug() << endl;
  
  char *str;
  hipHostMalloc(&str, 10000);

  checkPhrase<<<1,1>>>(str, *input);
  hipDeviceSynchronize();
  cerr << "checkId=" << str << endl;

  hipFree(str);

  //exit(0);
}


void Test1()
{
  thrust::host_vector<int> data(N), sought(N_SOUGHT);
  thrust::host_vector<bool> out;


  // fill the arrays 'a' and 'b' on the CPU
  for (int i=0; i<N; i++) {
    data[i] = i * 2;
  }

  sought[0] = 4;
  sought[1] = 5;

  // SET
  /*
  std::cerr << "SET:" << std::endl;
  Set<int> myset(data);
  myset.Find(out, sought);
  Print(std::cerr, out);

  bool found = myset.Find(3);
  bool found2 = myset.Find(4);
  std::cerr << "found:" << found << " " << found2 << std::endl;

  // INSERT
  std::cerr << "BEFORE:" << myset.Debug() << std::endl;
  myset.Insert(5);
  std::cerr << "AFTER:" << myset.Debug() << std::endl;

  // ERASE
  myset.Erase(14);
  std::cerr << "AFTER:" << myset.Debug() << std::endl;

  myset.Erase(11);
  std::cerr << "AFTER:" << myset.Debug() << std::endl;

	// MAP

  std::cerr << "MAP:" << std::endl;
  typedef thrust::pair<int, float> Pair;
	thrust::host_vector<Pair> dataMap(5);
	dataMap[0] = Pair(4, 2342.4f);
	dataMap[1] = Pair(6, 6756.77f);
	dataMap[2] = Pair(7, -34.3434);
	dataMap[3] = Pair(10, 43.33f);
	dataMap[4] = Pair(34545, -343.7675f);

 	Map<int, float, ComparePair<int, float> > mymap(dataMap);
  std::cerr << "BEFORE:" << mymap.Debug() << std::endl;

	found = mymap.FindMap(3);
  found2 = mymap.FindMap(4);
  std::cerr << "found:"
  		<< found << " "
  		<< found2 << " "
  		<< mymap.FindMap(5) << " "
  		<< mymap.FindMap(6) << " "
  		<< mymap.FindMap(7) << " "
  		<< mymap.FindMap(8) << " "
  		<< std::endl;

	mymap.Insert(9, -323.2);
	mymap.Insert(5, -999.2);

  std::cerr << "AFTER:" << mymap.Debug() << std::endl;
*/
}

//////////////////////////
template<typename T>
__global__
void Resize(Vector<T> &arr)
{
  thrust::pair<bool, size_t> found;
  found = arr.UpperBound(100);

}


void Test5()
{
  Set<int> s;

  s.Insert(4);
  s.Insert(3);
  s.Insert(5);
  s.Insert(2);

  cerr << "set2=" << s.Debug() << endl;
}

void Test6()
{
  Map<int, float> m;
  m.Insert(4, 5.45);
  m.Insert(6, 5654.34);
  m.Insert(3, 1.99);

  cerr << "map2=" << m.Debug() << endl;
}

void Test()
{
  //Test1();
  //Test2();
  //Test3();
  //Test4();
  //Test5();
  Test6();
}
