#include "hip/hip_runtime.h"
#include <iostream>
#include "Test.h"
#include "Managed.h"
#include "CUDA/Set.h"
#include "CUDA/Map.h"

using namespace std;

//////////////////////////////////////////////////////

class C2 : public Managed
{
public:
	int i;

	C2(int v)
	{
		i = v;
	}

	__device__ void Add(int v)
	{
		i += v;
	}

};

__global__ void Temp(C2 &o)
{
  o.Add(2);
}

__global__ void KernelC2(C2 &o)
{
  o.Add(3);
  Temp<<<2,1>>>(o);
}

void Test2()
{
  //KernelTCLass<<<1,1>>>();
  //KernelTAdd<<<1,1>>>(3);
  C2 *oCPU = new C2(6);
  hipDeviceSynchronize();
  cerr << "oCPU=" << oCPU->i << endl;

  KernelC2<<<1,1>>>(*oCPU);
  hipDeviceSynchronize();
  cerr << "oCPU=" << oCPU->i << endl;

  Temp<<<2,1>>>(*oCPU);
  //oCPU->Add<<<1,1>>>(4);

}

//////////////////////////////////////////////////////
#define N 10
#define N_SOUGHT 2

void Test()
{
  thrust::host_vector<int> data(N), sought(N_SOUGHT);
  thrust::host_vector<bool> out;


  // fill the arrays 'a' and 'b' on the CPU
  for (int i=0; i<N; i++) {
    data[i] = i * 2;
  }

  sought[0] = 4;
  sought[1] = 5;

  // SET
  std::cerr << "SET:" << std::endl;
  Set<int> myset(data);
  myset.Find(out, sought);
  Print(std::cerr, out);

  bool found = myset.Find(3);
  bool found2 = myset.Find(4);
  std::cerr << "found:" << found << " " << found2 << std::endl;

  // INSERT
  std::cerr << "BEFORE:" << myset.Debug() << std::endl;
  myset.Insert(5);
  std::cerr << "AFTER:" << myset.Debug() << std::endl;

  // ERASE
  myset.Erase(14);
  std::cerr << "AFTER:" << myset.Debug() << std::endl;

  myset.Erase(11);
  std::cerr << "AFTER:" << myset.Debug() << std::endl;

	// MAP

  std::cerr << "MAP:" << std::endl;
  typedef thrust::pair<int, float> Pair;
	thrust::host_vector<Pair> dataMap(5);
	dataMap[0] = Pair(4, 2342.4f);
	dataMap[1] = Pair(6, 6756.77f);
	dataMap[2] = Pair(7, -34.3434);
	dataMap[3] = Pair(10, 43.33f);
	dataMap[4] = Pair(34545, -343.7675f);

 	Map<int, float, ComparePair<int, float> > mymap(dataMap);
  std::cerr << "BEFORE:" << mymap.Debug() << std::endl;

	found = mymap.FindMap(3);
  found2 = mymap.FindMap(4);
  std::cerr << "found:"
  		<< found << " "
  		<< found2 << " "
  		<< mymap.FindMap(5) << " "
  		<< mymap.FindMap(6) << " "
  		<< mymap.FindMap(7) << " "
  		<< mymap.FindMap(8) << " "
  		<< std::endl;

	mymap.Insert(9, -323.2);
	mymap.Insert(5, -999.2);

  std::cerr << "AFTER:" << mymap.Debug() << std::endl;

  Test2();
}
