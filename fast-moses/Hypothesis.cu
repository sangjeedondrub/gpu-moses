#include "hip/hip_runtime.h"
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "Manager.h"
#include "TargetPhrase.h"

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:m_mgr(&mgr)
,m_bitmap(mgr.GetInput().size())
,m_scores(4)
{
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  m_mgr = &mgr;
  m_prevHypo = NULL;
  m_targetPhrase = NULL;

  m_bitmap.Init();
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const Range &range)
{
	m_mgr = &mgr;
	m_prevHypo = &prevHypo;
	m_targetPhrase = &tp;

	const Bitmap &prevBM = prevHypo.GetBitmap();
  m_bitmap.Init(prevBM, range);

  m_scores.PlusEqual(tp.GetScores());
  m_scores.PlusEqual(prevHypo.m_scores);
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  SCORE score = hypo.GetScores().GetTotal();
  output = score;
}

__host__
SCORE Hypothesis::GetFutureScore() const
{
  SCORE *output;
  hipMallocManaged(&output, sizeof(SCORE));
  getTotalScore<<<1,1>>>(*this, *output);
  SCORE score = *output;
  hipFree(output);

  return score;
}


