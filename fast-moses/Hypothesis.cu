#include "hip/hip_runtime.h"
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "Manager.h"
#include "TargetPhrase.h"

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:m_mgr(&mgr)
,m_bitmap(mgr.GetInput().size())
,m_scores(4)
{
   sss = 123.456;
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  m_mgr = &mgr;
  m_prevHypo = NULL;
  m_targetPhrase = NULL;

  m_bitmap.Init();
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const Range &range)
{
	m_mgr = &mgr;
	m_prevHypo = &prevHypo;
	m_targetPhrase = &tp;

	const Bitmap &prevBM = prevHypo.GetBitmap();
  m_bitmap.Init(prevBM, range);

  m_scores.PlusEqual(tp.GetScores());
  m_scores.PlusEqual(prevHypo.m_scores);
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  //SCORE score = hypo.GetScores().GetTotal();
  output = hypo.sss; //score;
  //output = 456.789;
}



