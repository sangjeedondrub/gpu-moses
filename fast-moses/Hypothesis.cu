#include "hip/hip_runtime.h"
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "TargetPhrase.h"
#include "Manager.h"
#include "System.h"

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:m_mgr(&mgr)
,m_bitmap(mgr.GetInput().size())
,m_scores(4)
{
   sss = mgr.system.ffs.totalSize;
   stateData = (char*) malloc(mgr.system.ffs.totalSize);
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  m_mgr = &mgr;
  m_prevHypo = NULL;
  m_targetPhrase = NULL;

  m_bitmap.Init();
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const Range &range)
{
	m_mgr = &mgr;
	m_prevHypo = &prevHypo;
	m_targetPhrase = &tp;

	const Bitmap &prevBM = prevHypo.GetBitmap();
  m_bitmap.Init(prevBM, range);

  m_scores.PlusEqual(tp.GetScores());
  m_scores.PlusEqual(prevHypo.m_scores);
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  output = hypo.GetScores().GetTotal();
  //output = hypo.sss; //score;
  //output = 456.789;
}

__host__
SCORE Hypothesis::GetFutureScore() const
{
    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*this, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    return h_s;
}

