#include "hip/hip_runtime.h"
#include <sstream>
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "TargetPhrase.h"
#include "Manager.h"
#include "System.h"
#include "CUDA/Util.h"

using namespace std;

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:mgr(&mgr)
,bitmap(mgr.GetInput().size())
,scores(mgr.system.featureFunctions.totalNumScores)
,stateData(mgr.system.featureFunctions.totalStateSize)
{
  sss = 453.54;
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  this->mgr = &mgr;
  prevHypo = NULL;

  path = &mgr.initPath;
  targetPhrase = &mgr.initPhrase;

  bitmap.Init();
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const InputPath &path)
{
  this->mgr = &mgr;
  this->prevHypo = &prevHypo;

  this->path = &path;
  targetPhrase = &tp;

  const Bitmap &prevBM = prevHypo.bitmap;
  bitmap.Init(prevBM, path.range);

  scores.PlusEqual(mgr.system, tp.GetScores());
  scores.PlusEqual(mgr.system, prevHypo.scores);

  mgr.system.featureFunctions.EvaluateWhenApplied(mgr, *this);
}

__device__
SCORE Hypothesis::getFutureScore() const
{
  return scores.GetTotal();
}

__device__
int Hypothesis::RecombineCompare(const Hypothesis &other) const
{
  // -1 = this < compare
  // +1 = this > compare
  // 0  = this ==compare
  int comp = bitmap.Compare(other.bitmap);
  if (comp != 0)
    return comp;

  return stateData.Compare(other.stateData);
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  output = hypo.getFutureScore();
  //output = hypo.sss; //score;
  //output = 456.789;
}

__host__
SCORE Hypothesis::GetFutureScore() const
{
    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*this, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    hipFree(d_s);

    return h_s;
}

///////////////////////////////////////////////////////////////
__host__
std::string Hypothesis::Debug() const
{
  stringstream strm;
  strm << this << " ";

  char *d_str;
  hipMallocManaged(&d_str, 1000);
  hipDeviceSynchronize();
  hipMemset(d_str, 0, 1000);

  debugObj<<<1,1>>>(*this, d_str);
  hipDeviceSynchronize();

  strm << d_str;

  hipFree(d_str);

  return strm.str();;
}

__device__
void Hypothesis::Debug(char *out) const
{
  path->range.Debug(out);
  StrCat(out, " ");
  bitmap.Debug(out);
  StrCat(out, " ");
  scores.Debug(out);

  if (prevHypo) {
    StrCat(out, "\n");
    prevHypo->Debug(out);
  }
}

