#include "hip/hip_runtime.h"
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "TargetPhrase.h"
#include "Manager.h"
#include "System.h"

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:mgr(&mgr)
,bitmap(mgr.GetInput().size())
,scores(4)
{
  sss = 453.54;
  stateData = (char*) malloc(mgr.system.ffs.totalSize);
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  this->mgr = &mgr;
  prevHypo = NULL;

  path = NULL;
  targetPhrase = NULL;

  bitmap.Init();
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const InputPath &path)
{
  this->mgr = &mgr;
  this->prevHypo = &prevHypo;

  this->path = &path;
  targetPhrase = &tp;

  const Bitmap &prevBM = prevHypo.bitmap;
  bitmap.Init(prevBM, path.range);

  scores.PlusEqual(tp.GetScores());
  scores.PlusEqual(prevHypo.scores);

  mgr.system.ffs.EvaluateWhenApplied(mgr, *this);
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  output = hypo.scores.GetTotal();
  //output = hypo.sss; //score;
  //output = 456.789;
}

__host__
SCORE Hypothesis::GetFutureScore() const
{
    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*this, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    return h_s;
}

