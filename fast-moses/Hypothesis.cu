#include "hip/hip_runtime.h"
#include "Hypothesis.h"
#include "Bitmap.h"
#include "Phrase.h"
#include "TargetPhrase.h"
#include "Manager.h"
#include "System.h"

__device__
Hypothesis::Hypothesis(const Manager &mgr)
:mgr(&mgr)
,bitmap(mgr.GetInput().size())
,scores(mgr.system, mgr.system.featureFunctions.totalNumScores)
,stateData(mgr.system.featureFunctions.totalStateSize)
{
  sss = 453.54;
}

  __device__
void Hypothesis::Init(const Manager &mgr)
{
  this->mgr = &mgr;
  prevHypo = NULL;

  path = &mgr.initPath;
  targetPhrase = &mgr.initPhrase;

  bitmap.Init();
}


__device__
void Hypothesis::Init(const Manager &mgr, const Hypothesis &prevHypo, const TargetPhrase &tp, const InputPath &path)
{
  this->mgr = &mgr;
  this->prevHypo = &prevHypo;

  this->path = &path;
  targetPhrase = &tp;

  const Bitmap &prevBM = prevHypo.bitmap;
  bitmap.Init(prevBM, path.range);

  scores.PlusEqual(mgr.system, tp.GetScores());
  scores.PlusEqual(mgr.system, prevHypo.scores);

  mgr.system.featureFunctions.EvaluateWhenApplied(mgr, *this);
}

__device__
SCORE Hypothesis::getFutureScore() const
{
  return scores.GetTotal();
}

__global__
void getTotalScore(const Hypothesis &hypo, SCORE &output)
{
  output = hypo.getFutureScore();
  //output = hypo.sss; //score;
  //output = 456.789;
}

__host__
SCORE Hypothesis::GetFutureScore() const
{
    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*this, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    hipFree(d_s);

    return h_s;
}

__device__
int Hypothesis::RecombineCompare(const Hypothesis &other) const
{
  // -1 = this < compare
  // +1 = this > compare
  // 0  = this ==compare
  int comp = bitmap.Compare(other.bitmap);
  if (comp != 0)
    return comp;

  return stateData.Compare(other.stateData);
}

