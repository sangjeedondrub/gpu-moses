#include "hip/hip_runtime.h"
#include "Distortion.h"
#include "../Hypothesis.h"
#include "../Range.h"

Distortion::Distortion()
{
	stateSize = sizeof(size_t);
}

__device__
void Distortion::EvaluateWhenApplied(const Manager &mgr, Hypothesis &hypo) const
{
  ScoresUnmanaged &scores = hypo.scores;

  //const Range &currRange = hypo.
}

__device__
int Distortion::ComputeDistortionDistance(const Range& prev,
    const Range& current) const
{
  int dist = 0;
  if (prev.GetNumWordsCovered() == 0) {
    dist = current.GetStartPos();
  }
  else {
    dist = (int) prev.GetEndPos() - (int) current.GetStartPos() + 1;
  }
  return abs(dist);
}
