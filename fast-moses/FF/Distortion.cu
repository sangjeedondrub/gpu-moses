#include "hip/hip_runtime.h"
#include "Distortion.h"
#include "../Hypothesis.h"
#include "../Range.h"
#include "../InputPath.h"

Distortion::Distortion()
{
	stateSize = sizeof(size_t);
	startInd = 4;
	numScores = 1;
}

__device__
void Distortion::EvaluateWhenApplied(const Manager &mgr, Hypothesis &hypo) const
{

  if (hypo.prevHypo) {
    const Range &prevRange = hypo.prevHypo->path->range;
    const Range &currRange = hypo.path->range;

    int dist = ComputeDistortionDistance(prevRange, currRange);

    ScoresUnmanaged &scores = hypo.scores;

    scores.PlusEqual(*this, currRange.startPos + currRange.endPos);
    //scores.PlusEqual(*this, dist);
  }

}

__device__
int Distortion::ComputeDistortionDistance(const Range& prev,
    const Range& current) const
{
  int dist = 0;
  if (prev.GetNumWordsCovered() == 0) {
    dist = current.startPos;
  }
  else {
    dist = (int) prev.endPos - (int) current.startPos + 1;
  }
  return abs(dist);
}
