#include "hip/hip_runtime.h"
#include "Distortion.h"
#include "../Hypothesis.h"
#include "../Range.h"
#include "../InputPath.h"

Distortion::Distortion()
{
	stateSize = sizeof(size_t);
}

__device__
void Distortion::EvaluateWhenApplied(const Manager &mgr, Hypothesis &hypo) const
{

  if (hypo.prevHypo) {
    const Range &prevRange = hypo.prevHypo->path->range;
    const Range &currRange = hypo.path->range;

    int dist = ComputeDistortionDistance(prevRange, currRange);

    ScoresUnmanaged &scores = hypo.scores;
    //scores.PlusEqual()
  }

}

__device__
int Distortion::ComputeDistortionDistance(const Range& prev,
    const Range& current) const
{
  int dist = 0;
  if (prev.GetNumWordsCovered() == 0) {
    dist = current.startPos;
  }
  else {
    dist = (int) prev.endPos - (int) current.startPos + 1;
  }
  return abs(dist);
}
