/*
 * FeatureFunction.h
 *
 *  Created on: 12 Dec 2016
 *      Author: hieu
 */
#include <vector>
#include <boost/foreach.hpp>
#include "FeatureFunctions.h"
#include "Distortion.h"
#include "WordPenalty.h"
#include "UnknownWordPenalty.h"
#include "../Parameter.h"
#include "../System.h"
#include "../TranslationModel/PhraseTableMemory.h"

using namespace std;


FeatureFunctions::FeatureFunctions(System &system)
:m_system(system)
,statelessFFs(0)
,statefulFFs(0)
{

}

FeatureFunctions::~FeatureFunctions()
{
  hipFree(pt);
  for (size_t i = 0; i < statelessFFs.size(); ++i) {
    hipFree(statelessFFs[i]);
  }
  for (size_t i = 0; i < statefulFFs.size(); ++i) {
    hipFree(statefulFFs[i]);
  }
}

__host__
void FeatureFunctions::Create()
{
  const Parameter &params = m_system.params;

  const PARAM_VEC *ffParams = params.GetParam("feature");
  UTIL_THROW_IF2(ffParams == NULL, "Must have [feature] section");

  totalNumScores = 0;
  totalStateSize = 0;

  BOOST_FOREACH(const std::string &line, *ffParams){
    cerr << "line=" << line << endl;
    vector<string> toks = Tokenize(line);
    assert(toks.size());

    FeatureFunction *ff;
    if (toks[0] == "Distortion") {
      ff = new Distortion();
    }
    else if (toks[0] == "WordPenalty") {
      ff = new WordPenalty();
    }
    else if (toks[0] == "PhraseDictionaryMemory") {
      ff = new PhraseTableMemory();
    }
    else if (toks[0] == "UnknownWordPenalty") {
      ff = new UnknownWordPenalty();
    }
    else {
      UTIL_THROW2("Unknown FF:" << line);
    }

    // put into correct vector
    assert(ff);
    ff->startInd = totalNumScores;
    totalNumScores += ff->numScores;

    StatefulFeatureFunction *sfff = dynamic_cast<StatefulFeatureFunction*>(ff);
    PhraseTableMemory *pt = dynamic_cast<PhraseTableMemory*>(ff);

    if (sfff) {
      sfff->stateOffset = totalStateSize;
      totalStateSize += sfff->stateSize;

      statefulFFs.PushBack(sfff);
    }
    else if (pt) {
      this->pt = pt;
    }
    else {
      statelessFFs.PushBack(ff);
    }


  }

  cerr << "statelessFFs=" << statelessFFs.size() << endl;
  cerr << "statefulFFs=" << statefulFFs.size() << endl;
  cerr << "totalNumScores=" << totalNumScores << endl;
  cerr << "totalStateSize=" << totalStateSize << endl;

}

__host__
void FeatureFunctions::Load()
{
  for (size_t i = 0; i < statelessFFs.size(); ++i) {
    statelessFFs[i]->Load(m_system);
  }
  for (size_t i = 0; i < statefulFFs.size(); ++i) {
    statefulFFs[i]->Load(m_system);
  }
  pt->Load(m_system);
}

__device__
void FeatureFunctions::EvaluateWhenApplied(const Manager &mgr, Hypothesis &hypo) const
{
  for (size_t i = 0; i < statefulFFs.size(); ++i) {
    const StatefulFeatureFunction *sfff = statefulFFs[i];

    switch (sfff->classId) {
    case FeatureFunction::ClassId::Distortion:
      const Distortion *castFF = static_cast<const Distortion*>(sfff);
      castFF->EvaluateWhenApplied(mgr, hypo);
      break;

    }
  }

}


