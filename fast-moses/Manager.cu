#include "hip/hip_runtime.h"
#include <iostream>
#include "Manager.h"
#include "Phrase.h"
#include "Hypothesis.h"
#include "PhraseTableMemory.h"

using namespace std;

Manager::Manager(const std::string &inputStr, const PhraseTableMemory &pt)
:m_pt(pt)
{
  m_input = Phrase::CreateFromString(inputStr);
  cerr << "m_input=" << m_input->Debug() << endl;

}

Manager::~Manager()
{
  delete m_input;
}

__global__ void checkManager(char *str, const Manager &mgr)
{
  const Phrase &input = mgr.GetInput();
  checkPhrase<<<1,1>>>(str, input);

}
///////////////////////////////////////
__global__ void Lookup(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  const Phrase &input = mgr.GetInput();
  size_t inputSize = input.size();

  const PhraseTableMemory &pt = mgr.GetPhraseTable();
  const TargetPhrases *tps = pt.Lookup(input, start, end);

  Array<const TargetPhrases*> &tpsArr = mgr.GetTargetPhrases();
  tpsArr[start * inputSize + end] = tps;
  //tpsArr[start * inputSize + end] = (const TargetPhrases*) 0x3434;

}

void Manager::Process()
{
  char *str;
  hipHostMalloc(&str, 10000);

  checkManager<<<1,1>>>(str, *this);
  hipDeviceSynchronize();
  cerr << "mgr=" << str << endl;

  size_t inputSize = m_input->GetSize();
  cerr << "inputSize=" << inputSize << endl;
  m_tpsArr.Resize(inputSize * inputSize, NULL);

  cerr << "before:" << DebugTPSArr() << endl;
  Lookup<<<inputSize, inputSize>>>(*this);
  hipDeviceSynchronize();
  cerr << "after:" << DebugTPSArr() << endl;

  /*
  m_stacks.Init(*this, m_input->GetSize() + 1);

  Hypothesis *hypo = new Hypothesis();
  Stack &stack = m_stacks[0];
  */
}

std::string Manager::DebugTPSArr() const
{
  std::stringstream strm;
  for (size_t i = 0; i < m_tpsArr.GetSize(); ++i) {
    strm << m_tpsArr[i] << " ";
  }
  return strm.str();
}
