#include "hip/hip_runtime.h"
#include <iostream>
#include "Manager.h"
#include "Phrase.h"
#include "Hypothesis.h"
#include "PhraseTableMemory.h"
#include "Stack.h"
#include "Range.h"

using namespace std;

Manager::Manager(const System &sys, const std::string &inputStr, const PhraseTableMemory &pt)
:system(sys)
,m_pt(pt)
,m_tpsVec(true, 0)
,initPhrase(0)
{
  m_input = Phrase::CreateFromString(inputStr);
  //cerr << "m_input=" << m_input->Debug() << endl;

}

Manager::~Manager()
{
  delete m_input;
}

__global__ void checkManager(char *str, const Manager &mgr)
{
  const Phrase &input = mgr.GetInput();
  checkPhrase<<<1,1>>>(str, input);

}


///////////////////////////////////////
__global__
void Lookup(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  const Phrase &input = mgr.GetInput();
  size_t inputSize = input.size();

  const PhraseTableMemory &pt = mgr.GetPhraseTable();
  const TargetPhrases *tps = pt.Lookup(input, start, end);

  mgr.GetInputPath(start, end).targetPhrases =  tps;
}

__global__
void Process1stStack(const Manager &mgr, Stacks &stacks)
{
  Hypothesis *hypo = new Hypothesis(mgr);
  hypo->Init(mgr);
  Stack &stack = stacks[0];
  stack.Add(hypo);
}

__global__
void ProcessStack(size_t stackInd, const Manager &mgr, Stacks &stacks)
{
  int hypoInd = blockIdx.x;
  int start = blockIdx.y;
  int end = blockIdx.z;

  if (start > end) {
    return;
  }

  const InputPath &path = mgr.GetInputPath(start, end);
  const TargetPhrases *tps = path.targetPhrases;
  if (tps == NULL || tps->size() == 0) {
    return;
  }

  const Stack &stack = stacks[stackInd];
  Hypothesis** vec = stack.GetArr();
  const Hypothesis &prevHypo = *vec[hypoInd];
  const Bitmap &prevBM = prevHypo.bitmap;

  if (prevBM.Overlap(path.range)) {
    return;
  }

  for (size_t i = 0; i < tps->size(); ++i) {
    const TargetPhrase *tp = (*tps)[i];
    assert(tp);

    Hypothesis *hypo = new Hypothesis(mgr);
    hypo->Init(mgr, prevHypo, *tp, path);
    const Bitmap &newBM = hypo->bitmap;
    size_t wordsCovered = newBM.GetNumWordsCovered();

    Stack &destStack = stacks[wordsCovered];

    Lock &lock = destStack.GetLock();
    lock.lock();

    destStack.Add(hypo);

    lock.unlock();
  }
}

///////////////////////////////////////
__host__
void Manager::Process()
{
  cerr << endl;
  /*
  char *str;
  hipHostMalloc(&str, 10000);

  checkManager<<<1,1>>>(str, *this);
  hipDeviceSynchronize();
  cerr << "mgr=" << str << endl;
  */

  size_t inputSize = m_input->GetSize();
  cerr << "inputSize=" << inputSize << endl;
  InitInputPaths();

  Lookup<<<inputSize, inputSize>>>(*this);
  hipDeviceSynchronize();
  cerr << "tps=" << DebugTPSArr() << endl;

  m_stacks.Init(*this, m_input->GetSize() + 1);

  Process1stStack<<<1,1>>>(*this, m_stacks);
  hipDeviceSynchronize();

  Stack &stack = m_stacks.Get(0);
  cerr << "1st stack=" << stack.GetSize() << endl;

  for (size_t stackInd = 0; stackInd < inputSize; ++stackInd) {
    const Stack &stack = m_stacks.Get(stackInd);
    size_t stackSize = stack.GetSize();

    //ProcessStack<<<1,1>>>(stackInd, *this, m_stacks);
    //ProcessStack<<<stackSize, 1>>>(stackInd, *this, m_stacks);
    //ProcessStack<<<1, inputSize>>>(stackInd, *this, m_stacks); // deadlock with lock
    //ProcessStack<<<stackSize, inputSize>>>(stackInd, *this, m_stacks);

    dim3 blocks(stackSize, inputSize, inputSize);
    ProcessStack<<<blocks, 1>>>(stackInd, *this, m_stacks);

    hipDeviceSynchronize();
    m_stacks.PrintStacks();
    //cerr << "stack=" << stack.Debug() << endl;
  }

  //cerr << "back=" << m_stacks.Back().Debug() << endl;

}

std::string Manager::DebugTPSArr() const
{
  std::stringstream strm;
  for (size_t i = 0; i < m_tpsVec.GetSize(); ++i) {
    const InputPath &path = m_tpsVec[i];
    const TargetPhrases *tps = path.targetPhrases;
    strm << path.range.Debug() << " ";
    strm << tps;
    if (tps) {
      strm << "(" << tps->GetSize() << ")";
    }
    strm << " " << endl;
  }
  return strm.str();
}

__global__
void InitPathRange(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  InputPath &path = mgr.GetInputPath(start, end);
  path.range = Range(start, end);
}

__host__
void Manager::InitInputPaths()
{
  size_t inputSize = m_input->GetSize();
  m_tpsVec.Resize(inputSize * inputSize);

  InitPathRange<<<inputSize, inputSize>>>(*this);

}

__device__
size_t Manager::RangeToInd(int start, int end) const
{
  const Phrase &input = GetInput();
  size_t inputSize = input.size();

  size_t ret = start * inputSize + end;
  return ret;
}

__device__
InputPath &Manager::GetInputPath(int start, int end)
{
  InputPath &path = m_tpsVec[RangeToInd(start, end)];
  return path;
}

__device__
const InputPath &Manager::GetInputPath(int start, int end) const
{
  const InputPath &path = m_tpsVec[RangeToInd(start, end)];
  return path;
}
