#include "hip/hip_runtime.h"
#include <iostream>
#include "Manager.h"
#include "Phrase.h"
#include "Hypothesis.h"
#include "PhraseTableMemory.h"
#include "Stack.h"

using namespace std;

Manager::Manager(const std::string &inputStr, const PhraseTableMemory &pt)
:m_pt(pt)
,m_tpsArr(true, 0)
{
  m_input = Phrase::CreateFromString(inputStr);
  //cerr << "m_input=" << m_input->Debug() << endl;

}

Manager::~Manager()
{
  delete m_input;
}

__global__ void checkManager(char *str, const Manager &mgr)
{
  const Phrase &input = mgr.GetInput();
  checkPhrase<<<1,1>>>(str, input);

}

///////////////////////////////////////
__global__ void Lookup(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  const Phrase &input = mgr.GetInput();
  size_t inputSize = input.size();

  const PhraseTableMemory &pt = mgr.GetPhraseTable();
  const TargetPhrases *tps = pt.Lookup(input, start, end);

  Array<const TargetPhrases*> &tpsArr = mgr.GetTargetPhrases();
  tpsArr[start * inputSize + end] = tps;
  //tpsArr[start * inputSize + end] = (const TargetPhrases*) 0x3434;

}

__global__ void Process1stStack(const Manager &mgr, Stacks &stacks)
{
  Hypothesis *hypo = new Hypothesis(mgr);
  hypo->Init(mgr);
  Stack &stack = stacks[0];
  stack.Add(hypo);
}

__global__ void ProcessStack(size_t stackInd, const Manager &mgr, Stacks &stacks)
{
  const Stack &stack = stacks[stackInd];

  int hypoInd = blockIdx.x;
  int start = threadIdx.x;

  const Set<Hypothesis*> &set = stack.GetSet();
  const Array<Hypothesis*> &vec = set.GetVec();
  const Hypothesis &prevHypo = *vec[hypoInd];

  Hypothesis *hypo = new Hypothesis(mgr);
  //hypo->Init(mgr, prevHypo);
  Stack &destStack = stacks[0];
  //destStack.Add(hypo);

}

///////////////////////////////////////

void Manager::Process()
{
  cerr << endl;
  /*
  char *str;
  hipHostMalloc(&str, 10000);

  checkManager<<<1,1>>>(str, *this);
  hipDeviceSynchronize();
  cerr << "mgr=" << str << endl;
  */

  size_t inputSize = m_input->GetSize();
  cerr << "inputSize=" << inputSize << endl;
  m_tpsArr.Resize(inputSize * inputSize, NULL);

  Lookup<<<inputSize, inputSize>>>(*this);
  hipDeviceSynchronize();
  cerr << "tps=" << DebugTPSArr() << endl;

  m_stacks.Init(*this, m_input->GetSize() + 1);

  Process1stStack<<<1,1>>>(*this, m_stacks);
  hipDeviceSynchronize();

  Stack &stack = m_stacks[0];
  cerr << "1st stack=" << stack.GetSize() << endl;

  for (size_t stackInd = 0; stackInd < inputSize; ++stackInd) {
    const Stack &stack = m_stacks[stackInd];
    size_t stackSize = stack.GetSize();

    ProcessStack<<<stackSize, inputSize>>>(stackInd, *this, m_stacks);
    hipDeviceSynchronize();
  }
}

std::string Manager::DebugTPSArr() const
{
  std::stringstream strm;
  for (size_t i = 0; i < m_tpsArr.GetSize(); ++i) {
    const TargetPhrases *tps = m_tpsArr[i];
    strm << tps;
    if (tps) {
      strm << "(" << tps->GetSize() << ")";
    }
    strm << " ";
  }
  return strm.str();
}
