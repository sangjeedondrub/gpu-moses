#include "hip/hip_runtime.h"
#include <iostream>
#include "Manager.h"
#include "Phrase.h"
#include "Hypothesis.h"
#include "PhraseTableMemory.h"
#include "Stack.h"
#include "Range.h"

using namespace std;

Manager::Manager(const std::string &inputStr, const PhraseTableMemory &pt)
:m_pt(pt)
,m_tpsVec(true, 0)
{
  m_input = Phrase::CreateFromString(inputStr);
  //cerr << "m_input=" << m_input->Debug() << endl;

}

Manager::~Manager()
{
  delete m_input;
}

__global__ void checkManager(char *str, const Manager &mgr)
{
  const Phrase &input = mgr.GetInput();
  checkPhrase<<<1,1>>>(str, input);

}

///////////////////////////////////////

__device__
const TargetPhrases *Manager::GetTargetPhrases(int start, int end) const
{
  const Phrase &input = GetInput();
  size_t inputSize = input.size();
  const TargetPhrases *tps = m_tpsVec[start * inputSize + end];
  return tps;
}

__device__
void Manager::SetTargetPhrases(int start, int end, const TargetPhrases *tps)
{
  const Phrase &input = GetInput();
  size_t inputSize = input.size();
  m_tpsVec[start * inputSize + end] = tps;
}

///////////////////////////////////////
__global__ void Lookup(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  const Phrase &input = mgr.GetInput();
  size_t inputSize = input.size();

  const PhraseTableMemory &pt = mgr.GetPhraseTable();
  const TargetPhrases *tps = pt.Lookup(input, start, end);

  mgr.SetTargetPhrases(start, end, tps);
}

__global__ void Process1stStack(const Manager &mgr, Stacks &stacks)
{
  Hypothesis *hypo = new Hypothesis(mgr);
  hypo->Init(mgr);
  Stack &stack = stacks[0];
  stack.Add(hypo);
}

__global__ void ProcessStack(size_t stackInd, const Manager &mgr, Stacks &stacks)
{
  int hypoInd = blockIdx.x;
  int start = blockIdx.y;
  int end = blockIdx.z;

  if (start > end) {
    return;
  }

  const Range range(start, end);

  const TargetPhrases *tps = mgr.GetTargetPhrases(start, end);
  if (tps == NULL || tps->size() == 0) {
    return;
  }

  const Stack &stack = stacks[stackInd];

  const Set<Hypothesis*> &set = stack.GetSet();
  const Vector<Hypothesis*> &vec = set.GetVec();
  const Hypothesis &prevHypo = *vec[hypoInd];
  const Bitmap &prevBM = prevHypo.GetBitmap();

  if (prevBM.Overlap(range)) {
    return;
  }

  for (size_t i = 0; i < tps->size(); ++i) {
    const TargetPhrase *tp = (*tps)[i];
    assert(tp);

    Hypothesis *hypo = new Hypothesis(mgr);
    hypo->Init(mgr, prevHypo, *tp, range);
    const Bitmap &newBM = hypo->GetBitmap();
    size_t wordsCovered = newBM.GetNumWordsCovered();

    Stack &destStack = stacks[wordsCovered];

    Lock &lock = destStack.GetLock();
    lock.lock();

    destStack.Add(hypo);

    lock.unlock();
  }
}

///////////////////////////////////////

void Manager::Process()
{
  cerr << endl;
  /*
  char *str;
  hipHostMalloc(&str, 10000);

  checkManager<<<1,1>>>(str, *this);
  hipDeviceSynchronize();
  cerr << "mgr=" << str << endl;
  */

  size_t inputSize = m_input->GetSize();
  cerr << "inputSize=" << inputSize << endl;
  m_tpsVec.Resize(inputSize * inputSize, NULL);

  Lookup<<<inputSize, inputSize>>>(*this);
  hipDeviceSynchronize();
  cerr << "tps=" << DebugTPSArr() << endl;

  m_stacks.Init(*this, m_input->GetSize() + 1);

  Process1stStack<<<1,1>>>(*this, m_stacks);
  hipDeviceSynchronize();

  Stack &stack = m_stacks[0];
  cerr << "1st stack=" << stack.GetSize() << endl;

  for (size_t stackInd = 0; stackInd < inputSize; ++stackInd) {
    const Stack &stack = m_stacks[stackInd];
    size_t stackSize = stack.GetSize();

    //ProcessStack<<<1,1>>>(stackInd, *this, m_stacks);
    //ProcessStack<<<stackSize, 1>>>(stackInd, *this, m_stacks);
    //ProcessStack<<<1, inputSize>>>(stackInd, *this, m_stacks); // deadlock with lock
    //ProcessStack<<<stackSize, inputSize>>>(stackInd, *this, m_stacks);

    dim3 blocks(stackSize, inputSize, inputSize);
    ProcessStack<<<blocks, 1>>>(stackInd, *this, m_stacks);

    hipDeviceSynchronize();
    m_stacks.PrintStacks();
  }
}

std::string Manager::DebugTPSArr() const
{
  std::stringstream strm;
  for (size_t i = 0; i < m_tpsVec.GetSize(); ++i) {
    const TargetPhrases *tps = m_tpsVec[i];
    strm << tps;
    if (tps) {
      strm << "(" << tps->GetSize() << ")";
    }
    strm << " ";
  }
  return strm.str();
}
