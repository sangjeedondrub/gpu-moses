#include "hip/hip_runtime.h"
#include <iostream>
#include "Manager.h"
#include "Phrase.h"
#include "Hypothesis.h"
#include "PhraseTableMemory.h"

using namespace std;

Manager::Manager(const std::string &inputStr, const PhraseTableMemory &pt)
:m_pt(pt)
{
  m_input = Phrase::CreateFromString(inputStr);
  cerr << "m_input=" << m_input->Debug() << endl;

}

Manager::~Manager()
{
  delete m_input;
}

__global__ void checkManager(char *str, const Manager &mgr)
{
  const Phrase &input = mgr.GetInput();
  checkPhrase<<<1,1>>>(str, input);

}
///////////////////////////////////////
__global__ void Lookup(Manager &mgr)
{
  int start = blockIdx.x;
  int end = threadIdx.x;

  if (start > end) {
    return;
  }

  const Phrase &input = mgr.GetInput();
  const PhraseTableMemory &pt = mgr.GetPhraseTable();
  pt.Lookup(input, start, end);


}

void Manager::Process()
{
  char *str;
  hipHostMalloc(&str, 10000);

  checkManager<<<1,1>>>(str, *this);
  hipDeviceSynchronize();
  cerr << "mgr=" << str << endl;

  size_t inputSize = m_input->GetSize();
  cerr << "inputSize=" << inputSize << endl;
  Lookup<<<inputSize, inputSize>>>(*this);

  /*
  m_stacks.Init(*this, m_input->GetSize() + 1);

  Hypothesis *hypo = new Hypothesis();
  Stack &stack = m_stacks[0];
  */
}


