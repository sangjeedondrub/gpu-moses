/*
 * PhraseTableMemory.cpp
 *
 *  Created on: 25 Nov 2016
 *      Author: hieu
 */
#include <iostream>
#include <string>
#include "PhraseTableMemory.h"
#include "MyVocab.h"
#include "InputFileStream.h"
#include "Util.h"
#include "../System.h"
#include "../FF/FeatureFunctions.h"

using namespace std;




/////////////////////////////////////////////////////////////////////////////////
PhraseTableMemory::PhraseTableMemory(size_t startInd, const std::string &line)
:FeatureFunction(startInd, line)
,m_root(NULL)
{
  classId = FeatureFunction::ClassId::PhraseDictionaryMemory;

  ReadParameters();

}

PhraseTableMemory::~PhraseTableMemory() {
	//delete m_root;
}

void PhraseTableMemory::Load(System &system)
{
	cerr << "begin loading" << endl;
	InputFileStream strm(m_path);

	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();

	std::string line;
	while (getline(strm, line)) {
		//cerr << line << endl;
		std::vector<std::string> toks;
		TokenizeMultiCharSeparator(toks, line, "|||");
		/*
		for (size_t i = 0; i < toks.size(); ++i) {
			cerr << "\t" << toks[i]<< endl;
		}
		*/

		vector<VOCABID> sourceIds = vocab.GetOrCreateIds(toks[0]);
		Phrase sourcePhrase(sourceIds);
		Node &node = m_root.AddOrCreateNode(sourceIds, NULL);

		/*
		cerr << "node=" << &node << " "
		    << node.GetChildren().GetSize() << " "
		    << endl;
    */
		TargetPhrase *tp = TargetPhrase::CreateFromString(system, toks[1]);
		tp->GetScores().CreateFromString(system, *this, toks[2], true);
		system.featureFunctions.EvaluateInIsolation(sourcePhrase, *tp);
		
    if (node.tps == NULL) {
      node.tps = new TargetPhrases();
    }
		node.tps->Add(tp);

		/*
		cerr << endl;
		cerr << "node=" << &node << " " << node.GetChildren().GetSize() << endl;
		cerr << "tp=" << tp->Debug() << endl;
    */

		/*
    char *str;
    hipHostMalloc(&str, 10000);

		checkPhrase<<<1,1>>>(str, *tp);
		hipDeviceSynchronize();
		//cerr << "totVocabId=" << str << endl;

		checkTargetPhrase<<<1,1>>>(str, *tp);
		hipDeviceSynchronize();
		//cerr << "totVocabId=" << str << endl;

		checkTargetPhrases<<<1,1>>>(str, tps);
		hipDeviceSynchronize();
    //cerr << "tps=" << str << endl;

    hipFree(str);
    */
	}

	//cerr << "root=" << m_root.GetChildren().Debug() << endl;
	cerr << "finished loading" << endl;
}

void PhraseTableMemory::SetParameter(const std::string& key, const std::string& value)
{
  if (key == "input-factor") {

  }
  else if (key == "output-factor") {

  }
  else if (key == "table-limit") {

  }
  else if (key == "path") {
    m_path = value;
  }
  else {
    FeatureFunction::SetParameter(key, value);
  }
}

__device__
const TargetPhrases *PhraseTableMemory::Lookup(const Phrase &phrase, size_t start, size_t end) const
{
  return m_root.Lookup(phrase, start, end, start);
}

