/*
 * PhraseTableMemory.cpp
 *
 *  Created on: 25 Nov 2016
 *      Author: hieu
 */
#include <iostream>
#include <string>
#include "PhraseTableMemory.h"
#include "MyVocab.h"
#include "InputFileStream.h"
#include "Util.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

using namespace std;

Node::Node()
:tps(NULL)
{}

Node::~Node()
{
  const Children::Vec &vec = m_children.GetVec();
  for (size_t i = 0; i < vec.size(); ++i) {
    const Children::Pair &pair = vec[i];
    const Node *node = pair.second;
    delete node;
  }
}

TargetPhrases &Node::GetTargetPhrases()
{
  if (tps == NULL) {
    tps = new TargetPhrases();
  }
  return *tps;
}

Node &Node::AddNode(const std::vector<VOCABID> &words, size_t pos)
{
	//cerr << "pos=" << pos << endl;
	if (pos >= words.size()) {
		//cerr << "found=" << pos << endl;
		return *this;
	}

	const Children::Vec &vec = m_children.GetVec();

	VOCABID vocabId = words[pos];
	Node *node;

	if (m_children.size()) {
		//bool exists = m_children.FindMap(vocabid);
		unsigned int ind = m_children.LowerBound(vocabId);

		if (ind < m_children.size()) {
			const Children::Pair &pair = vec[ind];
			VOCABID foundId = pair.first;
			if (foundId == vocabId) {
				node = pair.second;
			}
			else {
				node = new Node;
				m_children.Insert(vocabId, node);
			}
		}
		else {
			node = new Node;
			m_children.Insert(vocabId, node);
		}
	}
	else {
		node = new Node;
		m_children.Insert(vocabId, node);
	}

	node = &AddNode(words, pos + 1);
	return *node;
}

/////////////////////////////////////////////////////////////////////////////////
PhraseTableMemory::PhraseTableMemory() {
	// TODO Auto-generated constructor stub

}

PhraseTableMemory::~PhraseTableMemory() {
	// TODO Auto-generated destructor stub
}

void PhraseTableMemory::Load(const std::string &path)
{
	cerr << "begin loading" << endl;
	FastMoses::InputFileStream strm(path);

	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();

	std::string line;
	while (getline(strm, line)) {
		//cerr << line << endl;
		std::vector<std::string> toks;
		TokenizeMultiCharSeparator(toks, line, "|||");
		/*
		for (size_t i = 0; i < toks.size(); ++i) {
			cerr << "\t" << toks[i]<< endl;
		}
		*/

		vector<VOCABID> sourceIds = vocab.GetOrCreateIds(toks[0]);
		Node &node = m_root.AddNode(sourceIds);

		TargetPhrase *tp = TargetPhrase::CreateFromString(toks[1]);
		tp->GetScores().CreateFromString(toks[2]);
		
    TargetPhrases &tps = node.GetTargetPhrases();
		tps.Add(tp);

		/*
		cerr << endl << "tp=" << tp->Debug() << endl;

		VOCABID *totVocabId;
		hipHostMalloc(&totVocabId, sizeof(VOCABID));
		checkPhrase<<<1,1>>>(*totVocabId, *tp);
		hipDeviceSynchronize();
		cerr << "totVocabId=" << *totVocabId << endl;

		SCORE *totScore;
		hipHostMalloc(&totScore, sizeof(SCORE));
		checkTargetPhrase<<<1,1>>>(*totVocabId, *totScore, *tp);
		hipDeviceSynchronize();
		cerr << "totVocabId=" << *totVocabId << " " << *totScore << endl;
		 */

    size_t *tot;
    hipHostMalloc(&tot, sizeof(size_t));
		checkTargetPhrases<<<1,1>>>(*tot, tps);
		hipDeviceSynchronize();
    cerr << "tps=" << *tot << endl;

	}

	cerr << "finished loading" << endl;
}
