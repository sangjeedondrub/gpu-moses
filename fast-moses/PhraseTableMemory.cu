/*
 * PhraseTableMemory.cpp
 *
 *  Created on: 25 Nov 2016
 *      Author: hieu
 */
#include <iostream>
#include <string>
#include "PhraseTableMemory.h"
#include "MyVocab.h"
#include "InputFileStream.h"
#include "Util.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

using namespace std;

Node::Node()
:tps(NULL)
{}

Node::~Node()
{
  const Children::Vec &vec = m_children.GetVec();
  for (size_t i = 0; i < vec.size(); ++i) {
    const Children::Pair &pair = vec[i];
    const Node *node = pair.second;
    delete node;
  }
}

TargetPhrases &Node::GetTargetPhrases()
{
  if (tps == NULL) {
    tps = new TargetPhrases();
  }
  return *tps;
}

Node &Node::AddNode(const std::vector<VOCABID> &words, size_t pos)
{
	//cerr << "pos=" << pos << endl;
	if (pos >= words.size()) {
		//cerr << "found=" << pos << endl;
		return *this;
	}

	const Children::Vec &vec = m_children.GetVec();

	VOCABID vocabId = words[pos];
	Node *node;

	if (m_children.size()) {
		//bool exists = m_children.FindMap(vocabid);
		unsigned int ind = m_children.LowerBound(vocabId);

		if (ind < m_children.size()) {
			const Children::Pair &pair = vec[ind];
			VOCABID foundId = pair.first;
			if (foundId == vocabId) {
				node = pair.second;
			}
			else {
				node = new Node;
				m_children.Insert(vocabId, node);
			}
		}
		else {
			node = new Node;
			m_children.Insert(vocabId, node);
		}
	}
	else {
		node = new Node;
		m_children.Insert(vocabId, node);
	}

	node = &node->AddNode(words, pos + 1);
	return *node;
}

/////////////////////////////////////////////////////////////////////////////////
PhraseTableMemory::PhraseTableMemory()
{
  m_root = new Node();
}

PhraseTableMemory::~PhraseTableMemory() {
	delete m_root;
}

void PhraseTableMemory::Load(const std::string &path)
{
	cerr << "begin loading" << endl;
	FastMoses::InputFileStream strm(path);

	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();

	std::string line;
	while (getline(strm, line)) {
		//cerr << line << endl;
		std::vector<std::string> toks;
		TokenizeMultiCharSeparator(toks, line, "|||");
		/*
		for (size_t i = 0; i < toks.size(); ++i) {
			cerr << "\t" << toks[i]<< endl;
		}
		*/

		vector<VOCABID> sourceIds = vocab.GetOrCreateIds(toks[0]);
		Node &node = m_root->AddNode(sourceIds);

		TargetPhrase *tp = TargetPhrase::CreateFromString(toks[1]);
		tp->GetScores().CreateFromString(toks[2]);
		
    TargetPhrases &tps = node.GetTargetPhrases();
		tps.Add(tp);

		cerr << endl;
		cerr << "node=" << &node << " " << node.GetChildren().size() << endl;
		cerr << "tp=" << tp->Debug() << endl;

    char *str;
    hipHostMalloc(&str, 10000);

		checkPhrase<<<1,1>>>(str, *tp);
		hipDeviceSynchronize();
		cerr << "totVocabId=" << str << endl;

		checkTargetPhrase<<<1,1>>>(str, *tp);
		hipDeviceSynchronize();
		cerr << "totVocabId=" << str << endl;

		checkTargetPhrases<<<1,1>>>(str, tps);
		hipDeviceSynchronize();
    cerr << "tps=" << str << endl;

    hipFree(str);
	}

	cerr << "finished loading" << endl;
}
