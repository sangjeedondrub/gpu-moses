#include "hip/hip_runtime.h"
/*
 * PhraseTableMemory.cpp
 *
 *  Created on: 25 Nov 2016
 *      Author: hieu
 */
#include <iostream>
#include <string>
#include "PhraseTableMemory.h"
#include "MyVocab.h"
#include "InputFileStream.h"
#include "Util.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

using namespace std;

Node::~Node()
{
  const Children::Vec &vec = m_children.GetVec();
  for (size_t i = 0; i < vec.size(); ++i) {
    const Children::Pair &pair = vec[i];
    const Node *node = pair.second;
    delete node;
  }
}

Node &Node::AddNode(const std::vector<VOCABID> &words, size_t pos)
{
	//cerr << "pos=" << pos << endl;
	if (pos >= words.size()) {
		//cerr << "found=" << pos << endl;
		return *this;
	}

	const Children::Vec &vec = m_children.GetVec();

	VOCABID vocabId = words[pos];
	Node *node;

	if (m_children.size()) {
		//bool exists = m_children.FindMap(vocabid);
		unsigned int ind = m_children.LowerBound(vocabId);

		if (ind < m_children.size()) {
			const Children::Pair &pair = vec[ind];
			VOCABID foundId = pair.first;
			if (foundId == vocabId) {
				node = pair.second;
			}
			else {
				node = new Node;
				m_children.Insert(vocabId, node);
			}
		}
		else {
			node = new Node;
			m_children.Insert(vocabId, node);
		}
	}
	else {
		node = new Node;
		m_children.Insert(vocabId, node);
	}

	node = &AddNode(words, pos + 1);
	return *node;
}

/////////////////////////////////////////////////////////////////////////////////
PhraseTableMemory::PhraseTableMemory() {
	// TODO Auto-generated constructor stub

}

__global__ void checkTotalVocabId(VOCABID &totVocabId, const TargetPhrase *tp)
{
  size_t size = tp->size();
  totVocabId = size;
  for (size_t i = 0; i < size; ++i) {
    VOCABID id = (*tp)[i];
    totVocabId += id;
  }

  //hipMemcpy(&totVocabId, &sum, sizeof(VOCABID), hipMemcpyDeviceToHost);
}

PhraseTableMemory::~PhraseTableMemory() {
	// TODO Auto-generated destructor stub
}

void PhraseTableMemory::Load(const std::string &path)
{
	cerr << "begin loading" << endl;
	FastMoses::InputFileStream strm(path);

	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();

	std::string line;
	while (getline(strm, line)) {
		//cerr << line << endl;
		std::vector<std::string> toks;
		TokenizeMultiCharSeparator(toks, line, "|||");
		/*
		for (size_t i = 0; i < toks.size(); ++i) {
			cerr << "\t" << toks[i]<< endl;
		}
		*/

		vector<VOCABID> sourceIds = vocab.GetOrCreateIds(toks[0]);
		Node &node = m_root.AddNode(sourceIds);

		TargetPhrase *tp = TargetPhrase::CreateFromString(toks[1]);
		tp->GetScores().CreateFromString(toks[2]);
		cerr << "tp=" << tp->Debug() << endl;

		hipDeviceSynchronize();
		VOCABID *totVocabId;
		hipHostMalloc(&totVocabId, sizeof(VOCABID));
		checkTotalVocabId<<<1,1>>>(*totVocabId, tp);
		cerr << "totVocabId=" << *totVocabId << endl;

		node.GetTargetPhrases().Add(tp);

	}

	cerr << "finished loading" << endl;
}
