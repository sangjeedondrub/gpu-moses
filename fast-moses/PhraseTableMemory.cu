/*
 * PhraseTableMemory.cpp
 *
 *  Created on: 25 Nov 2016
 *      Author: hieu
 */
#include <iostream>
#include <string>
#include "PhraseTableMemory.h"
#include "MyVocab.h"
#include "InputFileStream.h"
#include "Util.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

using namespace std;

Node::Node()
:m_tps(NULL)
{}

Node::~Node()
{
  const Children::Vec &vec = m_children.GetVec();
  for (size_t i = 0; i < vec.GetSize(); ++i) {
    const Children::Pair &pair = vec[i];
    const Node *node = pair.second;
    delete node;
  }
}

TargetPhrases &Node::GetTargetPhrases()
{
  if (m_tps == NULL) {
    hipMallocManaged(&m_tps, sizeof(TargetPhrases));
  }
  return *m_tps;
}

__host__
Node &Node::AddNode(const std::vector<VOCABID> &words, size_t pos)
{
	//cerr << "pos=" << pos << endl;
	if (pos >= words.size()) {
		//cerr << "found=" << pos << endl;
		return *this;
	}

	VOCABID vocabId = words[pos];
	Node *node;

	thrust::pair<bool, size_t> upper = m_children.UpperBound(vocabId);
	if (upper.first) {
	  size_t ind = upper.second;
	  node = m_children.GetValue(ind);
	}
	else {
	  hipMallocManaged(&node, sizeof(Node));
    m_children.Insert(vocabId, node);
	}

	node = &node->AddNode(words, pos + 1);
	return *node;
}

__device__
const TargetPhrases *Node::Lookup(const Phrase &phrase, size_t start, size_t end, size_t pos) const
{
  if (pos > end) {
    return m_tps;
  }

  VOCABID vocabId = phrase[pos];
  thrust::pair<bool, size_t> upper = m_children.UpperBound(vocabId);
  return (const TargetPhrases *) m_children.size();

  if (upper.first) {
    const Node *node = m_children.GetValue(upper.second);
    assert(node);
    return node->Lookup(phrase, start, end, pos + 1);
  }
  else {
    return (const TargetPhrases *) 0x987;
    return NULL;
  }
}

/////////////////////////////////////////////////////////////////////////////////
PhraseTableMemory::PhraseTableMemory()
{
  //m_root = new Node();
}

PhraseTableMemory::~PhraseTableMemory() {
	//delete m_root;
}

void PhraseTableMemory::Load(const std::string &path)
{
	cerr << "begin loading" << endl;
	FastMoses::InputFileStream strm(path);

	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();

	std::string line;
	while (getline(strm, line)) {
		//cerr << line << endl;
		std::vector<std::string> toks;
		TokenizeMultiCharSeparator(toks, line, "|||");
		/*
		for (size_t i = 0; i < toks.size(); ++i) {
			cerr << "\t" << toks[i]<< endl;
		}
		*/

		vector<VOCABID> sourceIds = vocab.GetOrCreateIds(toks[0]);
		Node &node = m_root.AddNode(sourceIds);

		TargetPhrase *tp = TargetPhrase::CreateFromString(toks[1]);
		tp->GetScores().CreateFromString(toks[2]);
		
    TargetPhrases &tps = node.GetTargetPhrases();
		tps.Add(tp);

		/*
		cerr << endl;
		cerr << "node=" << &node << " " << node.GetChildren().GetSize() << endl;
		cerr << "tp=" << tp->Debug() << endl;
    */

    char *str;
    hipHostMalloc(&str, 10000);

		checkPhrase<<<1,1>>>(str, *tp);
		hipDeviceSynchronize();
		//cerr << "totVocabId=" << str << endl;

		checkTargetPhrase<<<1,1>>>(str, *tp);
		hipDeviceSynchronize();
		//cerr << "totVocabId=" << str << endl;

		checkTargetPhrases<<<1,1>>>(str, tps);
		hipDeviceSynchronize();
    //cerr << "tps=" << str << endl;

    hipFree(str);
	}

	cerr << "root=" << m_root.GetChildren().Debug() << endl;
	cerr << "finished loading" << endl;
}

__device__
const TargetPhrases *PhraseTableMemory::Lookup(const Phrase &phrase, size_t start, size_t end) const
{
  return m_root.Lookup(phrase, start, end, start);
}

