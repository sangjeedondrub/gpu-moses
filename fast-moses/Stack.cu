#include "hip/hip_runtime.h"
#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

__global__
void InitStack(Stack &stack)
{
}


Stack::Stack()
:m_coll()
{
  m_coll.GetVec().Reserve(5000);

  hipDeviceSynchronize();
  //cerr << "m_arr=" << m_arr << endl;
}

__host__
Stack::~Stack()
{
  for (size_t i = 0; i < m_coll.size(); ++i) {
    Hypothesis *hypo = m_coll.GetVec()[i];
    hipFree(hypo);
  }
}

__device__
void Stack::add(Hypothesis *hypo)
{
	m_coll.insert(hypo);
  //(*m_arr)[m_size] = hypo;
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = m_coll.GetVec()[i];
    cerr << "HH2:" << hypo << endl;

    SCORE h_s;
    h_s = hypo->GetFutureScore();

    cerr << "HH3:" << h_s << endl;
  }

  return strm.str();
}
