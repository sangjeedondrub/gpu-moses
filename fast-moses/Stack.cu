#include "hip/hip_runtime.h"
#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

Stack::Stack()
:m_coll(false)
{}

  __device__
void Stack::Add(Hypothesis *hypo)
{
	m_coll.insert(hypo);
}

__global__
void getTotalScore(const Hypothesis *hypo)
{

}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  strm << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1" << endl;
    const Hypothesis *hypo = m_coll.GetVec()[i];
    cerr << "before" << endl;
    getTotalScore<<<1,1>>>(hypo);
    cerr << "after" << endl;

    //strm << Hypothesis::GetTotalScore(hypo) << " ";
    //strm << (size_t) hypo << " ";
    //strm << i << " ";
  }

  return strm.str();
}
