#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;


Stack::Stack()
:m_coll()
{
  m_coll.GetVec().Reserve(5000);

  hipDeviceSynchronize();
  //cerr << "m_arr=" << m_arr << endl;
}

__host__
Stack::~Stack()
{
  for (size_t i = 0; i < m_coll.size(); ++i) {
    Hypothesis *hypo = m_coll.GetVec()[i];
    hipFree(hypo);
  }
}

__device__
void Stack::add(Hypothesis *hypo)
{
  thrust::pair<bool, size_t> upper = m_coll.upperBound(hypo);
  if (upper.first) {
    // same hypo exist
    const Hypothesis *otherHypo = m_coll.GetVec()[upper.second];

    SCORE newScore = hypo->getFutureScore();
    SCORE otherScore = otherHypo->getFutureScore();

    if (newScore > otherScore) {
      // new hypo is better
      delete otherHypo;
      m_coll.GetVec()[upper.second] = hypo;
    }
    else {
      // existing hypo is better
      delete hypo;
    }
  }
  else {
    m_coll.insert(hypo);
    //(*m_arr)[m_size] = hypo;
  }
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = m_coll.GetVec()[i];
    cerr << "HH2:" << hypo << endl;

    SCORE h_s;
    h_s = hypo->GetFutureScore();

    cerr << "HH3:" << h_s << endl;
    cerr << "HH4:" << hypo->Debug() << endl;
  }

  return strm.str();
}
