#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

Stack::Stack()
:m_coll(false)
{}

  __device__
void Stack::Add(Hypothesis *hypo)
{
	m_coll.insert(hypo);
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  strm << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = m_coll.GetVec().Get(i);
    cerr << "HH2:" << hypo << endl;

    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*hypo, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    cerr << "HH3:" << h_s << endl;
    hipFree(d_s);

    //getTotalScore<<<1,1>>>(hypo);
    //strm << hypo->GetFutureScore() << " ";
    //cerr << "HH3:" << hypo->GetFutureScore() << endl;
    //strm << Hypothesis::GetTotalScore(hypo) << " ";
    //strm << (size_t) hypo << " ";
    //strm << i << " ";
  }

  return strm.str();
}
