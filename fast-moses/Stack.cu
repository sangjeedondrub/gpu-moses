#include "hip/hip_runtime.h"
#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

__global__
void InitStack(Stack &stack)
{
}


Stack::Stack()
:m_arr(NULL)
{
  m_size = 0;

  Vector<Hypothesis*> *arr = new Vector<Hypothesis*>(true, 0);
  arr->Reserve(5000);
  m_arr = arr;

  hipDeviceSynchronize();
  //cerr << "m_arr=" << m_arr << endl;
}

__device__
void Stack::add(Hypothesis *hypo)
{
	//m_arr->push_back(hypo);
  (*m_arr)[m_size] = hypo;
	++m_size;
}

__host__
Hypothesis *Stack::Get(size_t ind) const
{
  Hypothesis *ret = m_arr->Get(ind);
  return ret;
  
	//return m_arr[ind];
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = Get(i);
    cerr << "HH2:" << hypo << endl;

    SCORE h_s;
    h_s = hypo->GetFutureScore();

    cerr << "HH3:" << h_s << endl;
  }

  return strm.str();
}
