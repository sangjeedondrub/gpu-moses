#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

Stack::Stack()
{
  hipMalloc(&m_arr, sizeof(Hypothesis*) * 5000);
  hipMemset(m_arr, 0, sizeof(Hypothesis*) * 5000);
  m_size = 0;
  //cerr << "m_arr=" << m_arr << endl;
}

__device__
void Stack::Add(Hypothesis *hypo)
{
	m_arr[m_size] = hypo;
	++m_size;
}

__host__
Hypothesis *Stack::Get(size_t ind) const
{
  Hypothesis *ret;
  //hipMalloc(&ret, sizeof(Hypothesis*));

  hipMemcpy(&ret, &m_arr[ind], sizeof(Hypothesis *), hipMemcpyDeviceToHost);
  return ret;
  
	//return m_arr[ind];
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = Get(i);
    cerr << "HH2:" << hypo << endl;

    SCORE *d_s;
    hipMalloc(&d_s, sizeof(SCORE));
    hipDeviceSynchronize();

    getTotalScore<<<1,1>>>(*hypo, *d_s);
    hipDeviceSynchronize();
    
    SCORE h_s;
    hipMemcpy(&h_s, d_s, sizeof(SCORE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    cerr << "HH3:" << h_s << endl;
    hipFree(d_s);

    //getTotalScore<<<1,1>>>(hypo);
    //strm << hypo->GetFutureScore() << " ";
    //cerr << "HH3:" << hypo->GetFutureScore() << endl;
    //strm << Hypothesis::GetTotalScore(hypo) << " ";
    //strm << (size_t) hypo << " ";
    //strm << i << " ";
  }

  return strm.str();
}
