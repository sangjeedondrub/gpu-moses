#include "hip/hip_runtime.h"
#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

__global__
void InitStack(Stack &stack)
{
  Array<Hypothesis*> *arr = new Array<Hypothesis*>(5000);
  stack.m_arr = arr;
}


Stack::Stack()
:m_arr(NULL)
{
  m_size = 0;
  InitStack<<<1,1>>>(*this);
  hipDeviceSynchronize();
  //cerr << "m_arr=" << m_arr << endl;
}

__device__
void Stack::Add(Hypothesis *hypo)
{
	(*m_arr)[m_size] = hypo;
	++m_size;
}

__host__
Hypothesis *Stack::Get(size_t ind) const
{
  Hypothesis *ret = m_arr->Get(ind);
  return ret;
  
	//return m_arr[ind];
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = Get(i);
    cerr << "HH2:" << hypo << endl;

    SCORE h_s;
    h_s = hypo->GetFutureScore();

    cerr << "HH3:" << h_s << endl;
  }

  return strm.str();
}
