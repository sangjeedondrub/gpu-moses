#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

Stack::Stack()
{
  hipMalloc(&m_arr, sizeof(Hypothesis*) * 5000);
  hipMemset(m_arr, 0, sizeof(Hypothesis*) * 5000);
  m_size = 0;
  //cerr << "m_arr=" << m_arr << endl;
}

__device__
void Stack::Add(Hypothesis *hypo)
{
	m_arr[m_size] = hypo;
	++m_size;
}

__host__
Hypothesis *Stack::Get(size_t ind) const
{
  Hypothesis *ret;
  //hipMalloc(&ret, sizeof(Hypothesis*));

  hipMemcpy(&ret, &m_arr[ind], sizeof(Hypothesis *), hipMemcpyDeviceToHost);
  return ret;
  
	//return m_arr[ind];
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = Get(i);
    cerr << "HH2:" << hypo << endl;

    SCORE h_s;
		h_s = hypo->GetFutureScore();

    cerr << "HH3:" << h_s << endl;

    //getTotalScore<<<1,1>>>(hypo);
    //strm << hypo->GetFutureScore() << " ";
    //cerr << "HH3:" << hypo->GetFutureScore() << endl;
    //strm << Hypothesis::GetTotalScore(hypo) << " ";
    //strm << (size_t) hypo << " ";
    //strm << i << " ";
  }

  return strm.str();
}
