#include "hip/hip_runtime.h"
#include <iostream>
#include "Stack.h"
#include "Hypothesis.h"

using namespace std;

__global__
void InitStack(Stack &stack)
{
}


Stack::Stack()
:m_arr(0)
{
  m_size = 0;

  m_arr.Reserve(5000);

  hipDeviceSynchronize();
  //cerr << "m_arr=" << m_arr << endl;
}

__host__
Stack::~Stack()
{
  for (size_t i = 0; i < m_arr.size(); ++i) {
    Hypothesis *hypo = m_arr[i];
    hipFree(hypo);
  }
}

__device__
void Stack::add(Hypothesis *hypo)
{
	m_arr.push_back(hypo);
  //(*m_arr)[m_size] = hypo;
	++m_size;
}

__host__
Hypothesis *Stack::Get(size_t ind) const
{
  Hypothesis *ret = m_arr[ind];
  return ret;
  
	//return m_arr[ind];
}

__host__
std::string Stack::Debug() const
{
  std::stringstream strm;
  size_t size = GetSize();
  cerr << "size=" << size << ":";
  for (size_t i = 0; i < size; ++i) {
    cerr << "HH1:" << i << endl;
    const Hypothesis *hypo = Get(i);
    cerr << "HH2:" << hypo << endl;

    SCORE h_s;
    h_s = hypo->GetFutureScore();

    cerr << "HH3:" << h_s << endl;
  }

  return strm.str();
}
