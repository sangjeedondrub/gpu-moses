#include "hip/hip_runtime.h"
#include <sstream>
#include <vector>
#include "Phrase.h"
#include "MyVocab.h"

using namespace std;

Phrase *Phrase::CreateFromString(const std::string &str)
{
	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();
	vector<VOCABID> ids = vocab.GetOrCreateIds(str);
	Phrase *tp = new Phrase(ids);
	return tp;
}

__host__ Phrase::Phrase(const std::vector<VOCABID> &ids)
:m_vec(ids.size())
{
	//hipDeviceSynchronize();
	//cerr << "GetSize()=" << m_vec->GetSize() << endl;
	for (size_t i = 0; i < ids.size(); ++i) {
	    //cerr << i << "=" << ids[i] << endl;
  	    m_vec.Set(i, ids[i]);
	}
	//hipDeviceSynchronize();
}

__host__ std::string Phrase::Debug() const
{
  return m_vec.Debug();
}


__global__ void checkPhrase(VOCABID &totVocabId, const Phrase &phrase)
{
  size_t size = phrase.size();
  totVocabId = size;
  for (size_t i = 0; i < size; ++i) {
    VOCABID id = phrase[i];
    totVocabId += id;
  }

  //hipMemcpy(&totVocabId, &sum, sizeof(VOCABID), hipMemcpyDeviceToHost);                                             
}

