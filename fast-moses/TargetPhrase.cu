#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrase.h"
#include "MyVocab.h"

using namespace std;

TargetPhrase *TargetPhrase::CreateFromString(const std::string &str)
{
	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();
	vector<VOCABID> ids = vocab.GetOrCreateIds(str);
        cerr << "ids=" << ids.size() << endl;
	TargetPhrase *tp = new TargetPhrase(ids);
	return tp;
}

TargetPhrase::TargetPhrase(const std::vector<VOCABID> &ids)
:Phrase(ids)
,m_scores(4)
{
}

__host__ std::string TargetPhrase::Debug() const
{
  stringstream strm;
  strm << Phrase::Debug() << " Scores:" << m_scores.Debug();

  return strm.str();
}

__global__ void checkTargetPhrase(VOCABID &totVocabId, SCORE &totScore, const TargetPhrase &phrase)
{
  size_t size = phrase.size();
  totVocabId = size;
  for (size_t i = 0; i < size; ++i) {
    VOCABID id = phrase[i];
    totVocabId += id;
  }

  size = phrase.GetScores().size();
  totScore = size;
  for (size_t i = 0; i < size; ++i) {
      SCORE score = phrase.GetScores()[i];
      totScore += score;
  }

}

