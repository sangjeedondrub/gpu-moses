#include "hip/hip_runtime.h"
#include <sstream>
#include "TargetPhrase.h"
#include "MyVocab.h"
#include "CUDA/Util.h"

using namespace std;

TargetPhrase *TargetPhrase::CreateFromString(const std::string &str)
{
	FastMoses::MyVocab &vocab = FastMoses::MyVocab::Instance();
	vector<VOCABID> ids = vocab.GetOrCreateIds(str);
  //cerr << "ids=" << ids.size() << endl;

	TargetPhrase *tp = new TargetPhrase(ids);
	return tp;
}

TargetPhrase::TargetPhrase(const std::vector<VOCABID> &ids)
:Phrase(ids)
,m_scores(4)
{
}

__host__ std::string TargetPhrase::Debug() const
{
  stringstream strm;
  strm << Phrase::Debug() << " Scores:" << m_scores.Debug();

  return strm.str();
}

__global__ void checkTargetPhrase(char *str, const TargetPhrase &phrase)
{
  VOCABID totVocabId;
  SCORE totScore;

  size_t size = phrase.size();
  totVocabId = size;
  for (size_t i = 0; i < size; ++i) {
    VOCABID id = phrase[i];
    totVocabId += id;
  }

  size = phrase.GetScores().size();
  totScore = size;
  for (size_t i = 0; i < size; ++i) {
      SCORE score = phrase.GetScores()[i];
      totScore += score;
  }

  char *tmp = itoaDevice(totVocabId + totScore);
  StrCpy(str, tmp);

}

