/* Copyright (C) 1989, 1990, 1991, 1992 Free Software Foundation, Inc.
     Written by James Clark (jjc@jclark.com)

This file is part of groff.

groff is free software; you can redistribute it and/or modify it under
the terms of the GNU General Public License as published by the Free
Software Foundation; either version 2, or (at your option) any later
version.

groff is distributed in the hope that it will be useful, but WITHOUT ANY
WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
for more details.

You should have received a copy of the GNU General Public License along
with groff; see the file COPYING.  If not, write to the Free Software
Foundation, 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA. */
#include <string.h>
#include <hip/hip_runtime.h>

#define INT_DIGITS 19		/* enough for 64 bit integer */

__device__
char *itoaDevice(int i)
{
  /* Room for INT_DIGITS digits, - and '\0' */
  static char buf[INT_DIGITS + 2];
  char *p = buf + INT_DIGITS + 1;	/* points to terminating '\0' */
  if (i >= 0) {
    do {
      *--p = '0' + (i % 10);
      i /= 10;
    } while (i != 0);
    return p;
  }
  else {			/* i < 0 */
    do {
      *--p = '0' - (i % 10);
      i /= 10;
    } while (i != 0);
    *--p = '-';
  }
  return p;
}

__device__
size_t strlenDevice(const char *str)
{
  register const char *s;

  for (s = str; *s; ++s);
  return(s - str);
}


__device__ void MemCpy(char *dest, const char *src, size_t count)
{
  for (size_t i = 0; i < count; ++i) {
    dest[i] = src[i];
  }
}

__device__ void StrCpy(char *dest, const char *src)
{
  size_t len = strlenDevice(src);
  MemCpy(dest, src, len + 1);
}
