#include "hip/hip_runtime.h"
#include <iostream>
#include "Stacks.h"
#include "Stack.h"

using namespace std;

__global__
void InitStacks(Stacks &stacks)
{
  int stackInd = blockIdx.x;
  Stack &stack = stacks[stackInd];

  Array<Hypothesis*> *arr = new Array<Hypothesis*>(5000);
  stack.m_arr = arr;
}

void Stacks::Init(const Manager &mgr, size_t numStacks)
{
	m_vec.Resize(numStacks);
	for (size_t i = 0; i < numStacks; ++i) {
		m_vec[i] = new Stack();
	}
	InitStacks<<<numStacks, 1>>>(*this);
}

void Stacks::PrintStacks() const
{
  cerr << "stacks=";
  for (size_t i = 0; i < m_vec.GetSize(); ++i) {
    Stack *stack = m_vec.Get(i);
    cerr << stack->GetSize() << " ";
  }
  cerr << endl;
}
